
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"
#include "utils/math_functions.h"

namespace dragon {

namespace kernel {

/* <T = float32, Device = CUDA> */

template <typename T>
__global__ void _PRelu(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[0];
    }
}

template <typename T>
__global__ void _PReluNCHW(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = (idx / dim) % channels;
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[c];
    }
}

template <typename T>
__global__ void _PReluNHWC(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                x,
    const T*                w,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % channels;
        y[idx] = (x[idx] > 0) * x[idx] +
            (x[idx] < 0) * x[idx] * w[c];
    }
}

template<> void PRelu<float, CUDAContext>(
    const int               count,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            x,
    const float*            w,
    float*                  y,
    CUDAContext*            ctx) {
    if (channel_shared) {
        _PRelu
            <<< CUDA_BLOCKS(count), CUDA_THREADS,
                0, ctx->cuda_stream() >>>(
            count, channels, dim, x, w, y
        );
    } else {
        if (data_format == "NCHW") {
            _PReluNCHW
                <<< CUDA_BLOCKS(count), CUDA_THREADS,
                    0, ctx->cuda_stream() >>>(
                count, channels, dim, x, w, y
            );
        } else if (data_format == "NHWC") {
            _PReluNHWC
                <<< CUDA_BLOCKS(count), CUDA_THREADS,
                    0, ctx->cuda_stream() >>>(
                count, channels, dim, x, w, y
            );
        } else {
            LOG(FATAL) << "Unknown data format: " << data_format;
        }
    }
}

/* <T = float32, Device = CUDA> */

template <typename T>
__global__ void _PReluGrad(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * (
            (x[idx] > 0) + (x[idx] <= 0) * w[0]
        );
    }
}

template <typename T>
__global__ void _PReluGradNCHW(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = (idx / dim) % channels;
        dx[idx] = dy[idx] * (
            (x[idx] > 0) + (x[idx] <= 0) * w[c]
        );
    }
}

template <typename T>
__global__ void _PReluGradNHWC(
    const int               count,
    const int               channels,
    const int               dim,
    const T*                dy,
    const T*                x,
    const T*                w,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int c = idx % channels;
        dx[idx] = dy[idx] * ((x[idx] > 0) + (x[idx] <= 0) * w[c]);
    }
}

template<> void PReluGrad<float, CUDAContext>(
    const int               count,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            dy,
    const float*            x,
    const float*            w,
    float*                  dx,
    CUDAContext*            ctx) {
    if (channel_shared) {
        _PReluGrad
            <<< CUDA_BLOCKS(count), CUDA_THREADS,
                0, ctx->cuda_stream() >>>(
            count, channels, dim, dy, x, w, dx
        );
    } else {
        if (data_format == "NCHW") {
            _PReluGradNCHW
                <<< CUDA_BLOCKS(count), CUDA_THREADS,
                    0, ctx->cuda_stream() >>>(
                count, channels, dim, dy, x, w, dx
            );
        } else if (data_format == "NHWC") {
            _PReluGradNHWC
                <<< CUDA_BLOCKS(count), CUDA_THREADS,
                    0, ctx->cuda_stream() >>>(
                count, channels, dim, dy, x, w, dx
            );
        } else {
            LOG(FATAL) << "Unknown data format: " << data_format;
        }
    }
}

/* <T = float32, Device = CUDA> */

template <typename T>
__global__ void _PReluWGradBcast(
    const int               count,
    const int               rows,
    const int               row_offset,
    const T*                dy,
    const T*                x,
    T*                      bcast_dw) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        bcast_dw[idx] = dy[idx] * x[idx] * (x[idx] <= 0);
        for (int n = 1; n < rows; n++) {
            const int cur_idx = idx + n * row_offset;
            bcast_dw[idx] +=
                dy[cur_idx] * x[cur_idx] * (x[cur_idx] <= 0);
        }
    }
}

template<> void PReluWGrad<float, CUDAContext>(
    const int               rows,
    const int               row_offset,
    const int               channels,
    const int               dim,
    const bool              channel_shared,
    const string&           data_format,
    const float*            dy,
    const float*            x,
    const float*            multiplier,
    float*                  bcast_dw,
    float*                  dw,
    CUDAContext*            ctx) {
    auto cdim = channels * dim;
    _PReluWGradBcast
        <<< CUDA_BLOCKS(cdim), CUDA_THREADS,
            0, ctx->cuda_stream() >>>(
        cdim, rows, row_offset, dy, x, bcast_dw
    );
    if (channel_shared) {
        math::Dot(
            channels * dim,
            bcast_dw, multiplier,
            dw, ctx
        );
    } else {
        if (data_format == "NCHW") {
            math::Gemv(
                CblasNoTrans,
                channels, dim,
                1.f, bcast_dw, multiplier,
                0.f, dw, ctx
            );
        } else if (data_format == "NHWC") {
            math::Gemv(
                CblasTrans,
                dim, channels,
                1.f, bcast_dw, multiplier,
                0.f, dw, ctx
            );
        } else {
            LOG(FATAL) << "Unknown data format: " << data_format;
        }
    }
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA