
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/cast.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/* <T = ?, Device = CUDA> */

template <typename T>
__global__ void _Maximum(
    const int               nthreads,
    const T*                x1,
    const T*                x2,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        y[i] = max(x1[i], x2[i]);
    }
}

/* <T = float16, Device = CUDA> */

template<> __global__ void _Maximum<half>(
    const int               nthreads,
    const half*             x1,
    const half*             x2,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hgt(x1[i], x2[i]) ? x1[i] : x2[i];
#endif
    }
}

/* <T = ?, Device = CUDA> */

template <typename T>
__global__ void _BroadcastMaximum(
    const int               nthreads,
    const T*                x1,
    const T                 x2,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        y[i] = max(x1[i], x2);
    }
}

/* <T = float16, Device = CUDA> */

template<>  __global__ void _BroadcastMaximum<half>(
    const int               nthreads,
    const half*             x1,
    const half              x2,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hgt(x1[i], x2) ? x1[i] : x2;
#endif
    }
}

/* <T = ?, Device = CUDA> */

template <typename T>
__global__ void _MaximumGrad(
    const int               nthreads,
    const T*                x1,
    const T*                x2,
    const T*                dy,
    T*                      dx1,
    T*                      dx2) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        const bool dy_to_dx1 = x1[i] > x2[i];
        dx1[i] = dy_to_dx1 ? dy[i] : T(0);
        dx2[i] = dy_to_dx1 ? T(0) : dy[i];
    }
}

/* <T = float16, Device = CUDA> */

template<> __global__ void _MaximumGrad<half>(
    const int               nthreads,
    const half*             x1,
    const half*             x2,
    const half*             dy,
    half*                   dx1,
    half*                   dx2) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
        const bool dy_to_dx1 = __hgt(x1[i], x2[i]);
        dx1[i] = dy_to_dx1 ? dy[i] : __float2half(0.f);
        dx2[i] = dy_to_dx1 ? __float2half(0.f) : dy[i];
#endif
    }
}

/* <T = ?, Device = CUDA> */

template <typename T>
__global__ void _BroadcastMaximumGrad(
    const int               nthreads,
    const T*                x1,
    const T                 x2,
    const T*                dy,
    T*                      dx1,
    T*                      dx2) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        dx1[i] = (x1[i] > x2) ? dy[i] : T(0);
    }
}

/* <T = float16, Device = CUDA> */

template<> __global__ void _BroadcastMaximumGrad<half>(
    const int               nthreads,
    const half*             x1,
    const half              x2,
    const half*             dy,
    half*                   dx1,
    half*                   dx2) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
        dx1[i] = __hgt(x1[i], x2) ?
            dy[i] : __float2half(0.f);
#endif
    }
}

/* Kernel Launchers */

#define DEFINE_MAXIMUM_KERNEL_LAUNCHER(name, T, T2) \
    template <> void name<T, CUDAContext>( \
        const int               count, \
        const T*                x1, \
        const T2                x2, \
        T*                      y, \
        CUDAContext*            ctx) { \
        _##name \
            << < CUDA_BLOCKS(count), CUDA_THREADS, \
                 0, ctx->cuda_stream() >> >( \
            count, x1, x2, y \
        ); \
    }

#define DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(name, T, T2) \
    template <> void name<T, CUDAContext>( \
        const int               count, \
        const T*                x1, \
        const T2                x2, \
        const T*                dy, \
        T*                      dx1, \
        T*                      dx2, \
        CUDAContext*            ctx) { \
        _##name \
            << < CUDA_BLOCKS(count), CUDA_THREADS, \
                 0, ctx->cuda_stream() >> >( \
            count, x1, x2, dy, dx1, dx2 \
        ); \
    }

DEFINE_MAXIMUM_KERNEL_LAUNCHER(Maximum, int8_t, int8_t*);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(Maximum, uint8_t, uint8_t*);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(Maximum, int, int*);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(Maximum, int64_t, int64_t*);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(Maximum, float, float*);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(Maximum, double, double*);

DEFINE_MAXIMUM_KERNEL_LAUNCHER(BroadcastMaximum, int8_t, int8_t);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(BroadcastMaximum, uint8_t, uint8_t);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(BroadcastMaximum, int, int);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(BroadcastMaximum, int64_t, int64_t);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(BroadcastMaximum, float, float);
DEFINE_MAXIMUM_KERNEL_LAUNCHER(BroadcastMaximum, double, double);

DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(MaximumGrad, int8_t, int8_t*);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(MaximumGrad, uint8_t, uint8_t*);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(MaximumGrad, int, int*);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(MaximumGrad, int64_t, int64_t*);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(MaximumGrad, float, float*);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(MaximumGrad, double, double*);

DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(BroadcastMaximumGrad, int8_t, int8_t);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(BroadcastMaximumGrad, uint8_t, uint8_t);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(BroadcastMaximumGrad, int, int);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(BroadcastMaximumGrad, int64_t, int64_t);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(BroadcastMaximumGrad, float, float);
DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER(BroadcastMaximumGrad, double, double);

template <> void Maximum<float16, CUDAContext>(
    const int               count,
    const float16*          x1,
    const float16*          x2,
    float16*                y,
    CUDAContext*            ctx) {
    _Maximum \
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        count,
        reinterpret_cast<const half*>(x1),
        reinterpret_cast<const half*>(x2),
        reinterpret_cast<half*>(y)
    );
}

template <> void BroadcastMaximum<float16, CUDAContext>(
    const int               count,
    const float16*          x1,
    const float16           x2,
    float16*                y,
    CUDAContext*            ctx) {
    _BroadcastMaximum \
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        count,
        reinterpret_cast<const half*>(x1),
        cast::to<half>(x2),
        reinterpret_cast<half*>(y)
    );
}

template <> void MaximumGrad<float16, CUDAContext>(
    const int               count,
    const float16*          x1,
    const float16*          x2,
    const float16*          dy,
    float16*                dx1,
    float16*                dx2,
    CUDAContext*            ctx) {
    _MaximumGrad \
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        count,
        reinterpret_cast<const half*>(x1),
        reinterpret_cast<const half*>(x2),
        reinterpret_cast<const half*>(dy),
        reinterpret_cast<half*>(dx1),
        reinterpret_cast<half*>(dx2)
    );
}

template <> void BroadcastMaximumGrad<float16, CUDAContext>(
    const int               count,
    const float16*          x1,
    const float16           x2,
    const float16*          dy,
    float16*                dx1,
    float16*                dx2,
    CUDAContext*            ctx) {
    _BroadcastMaximumGrad \
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        count,
        reinterpret_cast<const half*>(x1),
        cast::to<half>(x2),
        reinterpret_cast<const half*>(dy),
        reinterpret_cast<half*>(dx1),
        reinterpret_cast<half*>(dx2)
    );
}

#undef DEFINE_MAXIMUM_KERNEL_LAUNCHER
#undef DEFINE_MAXIMUM_GRAD_KERNEL_LAUNCHER

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA