
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/cast.h"
#include "utils/op_kernel.h"
#include "utils/math_utils.h"
#include "utils/cub_device.h"
#include "utils/math_functions.h"

namespace dragon {

namespace kernel {

/* <Tx = ?, Ty = ?, Device = CUDA> */

template <typename Tx, typename Ty>
__global__ void _ColwiseMoments(
    const int                   rows,
    const int                   cols,
    const Tx*                   x,
    Ty*                         mean,
    Ty*                         var) {
    __shared__ typename BlockReduce<Ty>::TempStorage m_storage;
    __shared__ typename BlockReduce<Ty>::TempStorage v_storage;
    const Ty scale = Ty(1) / (Ty)cols;
    CUDA_2D_KERNEL_LOOP1(i, rows) {
        Ty m_val = 0, v_val = 0;
        CUDA_2D_KERNEL_LOOP2(j, cols) {
            const int xi = i * cols + j;
#if __CUDA_ARCH__ >= 350
            m_val += __ldg(x + xi);
            v_val += __ldg(x + xi) * __ldg(x + xi);
#else
            m_val += x[xi];
            v_val += x[xi] * x[xi];
#endif
        }
        m_val = BlockReduce<Ty>(m_storage).Sum(m_val);
        v_val = BlockReduce<Ty>(v_storage).Sum(v_val);
        if (threadIdx.x == 0) {
            const Ty mu = m_val * scale;
            mean[i] = mu;
            var[i] = v_val * scale - mu * mu;
        }
    }
}

template<> __global__ void _ColwiseMoments<half, float>(
    const int                   rows,
    const int                   cols,
    const half*                 x,
    float*                      mean,
    float*                      var) {
#if __CUDA_ARCH__ >= 530
    __shared__ typename BlockReduce<float>::TempStorage m_storage;
    __shared__ typename BlockReduce<float>::TempStorage v_storage;
    const float scale = 1.f / (float)cols;
    CUDA_2D_KERNEL_LOOP1(i, rows) {
        float m_val = 0.f, v_val = 0.f;
        CUDA_2D_KERNEL_LOOP2(j, cols) {
            const int xi = i * cols + j;
            m_val += __half2float(__ldg(x + xi));
            v_val += __half2float(__ldg(x + xi)) *
                     __half2float(__ldg(x + xi));
        }
        m_val = BlockReduce<float>(m_storage).Sum(m_val);
        v_val = BlockReduce<float>(v_storage).Sum(v_val);
        if (threadIdx.x == 0) {
            const float mu = m_val * scale;
            mean[i] = mu;
            var[i] = v_val * scale - mu * mu;
        }
    }
#endif
}

template <typename Tx, typename Ty>
__global__ void _RowwiseMoments(
    const int                   rows,
    const int                   cols,
    const Tx*                   x,
    Ty*                         mean,
    Ty*                         var) {
    __shared__ typename BlockReduce<Ty>::TempStorage m_storage;
    __shared__ typename BlockReduce<Ty>::TempStorage v_storage;
    const Ty scale = Ty(1) / (Ty)rows;
    CUDA_2D_KERNEL_LOOP1(i, cols) {
        Ty m_val = 0, v_val = 0;
        CUDA_2D_KERNEL_LOOP2(j, rows) {
            const int xi = j * cols + i;
#if __CUDA_ARCH__ >= 350
            m_val += __ldg(x + xi);
            v_val += __ldg(x + xi) * __ldg(x + xi);
#else
            m_val += x[xi];
            v_val += x[xi] * x[xi];
#endif
        }
        m_val = BlockReduce<Ty>(m_storage).Sum(m_val);
        v_val = BlockReduce<Ty>(v_storage).Sum(v_val);
        if (threadIdx.x == 0) {
            const Ty mu = m_val * scale;
            mean[i] = mu;
            var[i] = v_val * scale - mu * mu;
        }
    }
}

template<> __global__ void _RowwiseMoments<half, float>(
    const int                   rows,
    const int                   cols,
    const half*                 x,
    float*                      mean,
    float*                      var) {
#if __CUDA_ARCH__ >= 530
    __shared__ typename BlockReduce<float>::TempStorage m_storage;
    __shared__ typename BlockReduce<float>::TempStorage v_storage;
    const float scale = 1.f / (float)rows;
    CUDA_2D_KERNEL_LOOP1(i, cols) {
        float m_val = 0.f, v_val = 0.f;
        CUDA_2D_KERNEL_LOOP2(j, rows) {
            const int xi = j * cols + i;
            m_val += __half2float(__ldg(x + xi));
            v_val += __half2float(__ldg(x + xi)) *
                     __half2float(__ldg(x + xi));
        }
        m_val = BlockReduce<float>(m_storage).Sum(m_val);
        v_val = BlockReduce<float>(v_storage).Sum(v_val);
        if (threadIdx.x == 0) {
            const float mu = m_val * scale;
            mean[i] = mu;
            var[i] = v_val * scale - mu * mu;
        }
    }
#endif
}

#define FIXED_DIVISOR_DIV_MOD(d, n, q, r) \
    do {                                  \
        const auto n_copy = n;            \
        *q = n_copy / d;                  \
        *r = n_copy % d;                  \
    } while (0)

template <typename Tx, typename Ty>
__global__ void _GenericMoments(
    const int                   ndims,
    const int                   outer_dim,
    const int                   inner_dim,
    const int*                  x_strides,
    const int*                  y_dims,
    const Tx*                   x,
    Ty*                         mean,
    Ty*                         var) {
    __shared__ typename BlockReduce<Ty>::TempStorage m_storage;
    __shared__ typename BlockReduce<Ty>::TempStorage v_storage;
    const Ty scale = Ty(1) / (Ty)inner_dim;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        Ty m_val = 0, v_val = 0;
        CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
            int xi = 0, yi = i * inner_dim + j;
#pragma unroll
            for (int d = ndims - 1; d >= 0; --d) {
                int r;
#if __CUDA_ARCH__ >= 350
                FIXED_DIVISOR_DIV_MOD(__ldg(y_dims + d), yi, &yi, &r);
                xi += r * __ldg(x_strides + d);
#else
                FIXED_DIVISOR_DIV_MOD(y_dims[d], yi, &yi, &r);
                xi += r * x_strides[d];
#endif
            }
#if __CUDA_ARCH__ >= 350
            m_val += __ldg(x + xi);
            v_val += __ldg(x + xi) * __ldg(x + xi);
#else
            m_val += x[xi];
            v_val += x[xi] * x[xi];
#endif
        }
        m_val = BlockReduce<Ty>(m_storage).Sum(m_val);
        v_val = BlockReduce<Ty>(v_storage).Sum(v_val);
        if (threadIdx.x == 0) {
            const Ty mu = m_val * scale;
            mean[i] = mu;
            var[i] = v_val * scale - mu * mu;
        }
    }
}

template<> __global__ void _GenericMoments<half, float>(
    const int                   ndims,
    const int                   outer_dim,
    const int                   inner_dim,
    const int*                  x_strides,
    const int*                  y_dims,
    const half*                 x,
    float*                      mean,
    float*                      var) {
#if __CUDA_ARCH__ >= 530
    __shared__ typename BlockReduce<float>::TempStorage m_storage;
    __shared__ typename BlockReduce<float>::TempStorage v_storage;
    const float scale = 1.f / (float)inner_dim;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        float m_val = 0.f, v_val = 0.f;
        CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
            int xi = 0, yi = i * inner_dim + j;
#pragma unroll
            for (int d = ndims - 1; d >= 0; --d) {
                int r;
                FIXED_DIVISOR_DIV_MOD(__ldg(y_dims + d), yi, &yi, &r);
                xi += r * __ldg(x_strides + d);
            }
            m_val += __half2float(__ldg(x + xi));
            v_val += __half2float(__ldg(x + xi)) *
                     __half2float(__ldg(x + xi));
        }
        m_val = BlockReduce<float>(m_storage).Sum(m_val);
        v_val = BlockReduce<float>(v_storage).Sum(v_val);
        if (threadIdx.x == 0) {
            const float mu = m_val * scale;
            mean[i] = mu;
            var[i] = v_val * scale - mu * mu;
        }
    }
#endif
}

template <typename Tx, typename Ty>
void _Moments(
    const int               ndims,
    const int*              dims,
    const int               naxes,
    const int*              axes,
    const Tx*               x,
    Ty*                     mean,
    Ty*                     var,
    CUDAContext*            ctx) {
    vec32_t y_dimsV(dims, dims + ndims);
    for (int i = 0; i < naxes; ++i) y_dimsV[axes[i]] = 1;
    const int* x_dims = dims; const int* y_dims = y_dimsV.data();
    const int x_size = std::accumulate(x_dims,
        x_dims + ndims, 1, std::multiplies<int>());
    const int y_size = std::accumulate(y_dims,
        y_dims + ndims, 1, std::multiplies<int>());

    int rows, cols;

    /*! Case #1: Colwise Reduce */
    if (utils::IsColwiseReduce(
            ndims, x_dims, y_dims,
                &rows, &cols)) {
        _ColwiseMoments
            <<< CUDA_2D_BLOCKS(rows), CUDA_THREADS,
                0, ctx->cuda_stream() >>>(
            rows, cols, x, mean, var
        ); return;
    }

    /*! Case #2: Rowwise Reduce */
    if (utils::IsRowwiseReduce(
            ndims, x_dims, y_dims, 
                &rows, &cols)) {
        _RowwiseMoments
            <<< CUDA_2D_BLOCKS(cols), CUDA_THREADS,
                0, ctx->cuda_stream() >>>(
            rows, cols, x, mean, var
        ); return;
    }

    /*! Case #3: Generic Reduce */
    vec32_t axesT(ndims), stridesT(ndims), dimsT(ndims);

    utils::ComputeTransposedAxesForReduce(
        ndims, naxes, axes,
        axesT.data()
   );

    utils::ComputeTransposedStrides(
        ndims, dims,
        axesT.data(),
        stridesT.data()
    );

    int outer_dim = 1, inner_dim = 1;
    const int pivot = ndims - naxes;
    for (int i = 0; i < pivot; ++i) outer_dim *= dims[axesT[i]];
    for (int i = pivot; i < ndims; ++i) inner_dim *= dims[axesT[i]];
    for (int i = 0; i < ndims; ++i) dimsT[i] = dims[axesT[i]];

    const size_t dbytes = sizeof(int) * ndims;
    int* XSS = (int*)ctx->New(dbytes), *YDS = (int*)ctx->New(dbytes);
    ctx->Memcpy<CUDAContext, CPUContext>(dbytes, XSS, stridesT.data());
    ctx->Memcpy<CUDAContext, CPUContext>(dbytes, YDS, dimsT.data());

    _GenericMoments
        <<< CUDA_2D_BLOCKS(outer_dim), CUDA_THREADS,
            0, ctx->cuda_stream() >>>(
        ndims, outer_dim, inner_dim,
        XSS, YDS, x, mean, var
    );

    ctx->FinishDeviceCompution();
    ctx->Delete(XSS); ctx->Delete(YDS);
}

/* Kernel Launchers */

#define DEFINE_MOMENTS_KERNEL_LAUNCHER(Tx, Ty) \
    template <> void Moments<Tx, Ty, CUDAContext>( \
        const int               ndims, \
        const int*              dims, \
        const int               naxes, \
        const int*              axes, \
        const Tx*               x, \
        Ty*                     mean, \
        Ty*                     var, \
        CUDAContext*            ctx) { \
        _Moments( \
            ndims, dims, \
            naxes, axes, \
            x, mean, var, ctx \
        ); \
    }

DEFINE_MOMENTS_KERNEL_LAUNCHER(int8_t, float);
DEFINE_MOMENTS_KERNEL_LAUNCHER(uint8_t, float);
DEFINE_MOMENTS_KERNEL_LAUNCHER(int, float);
DEFINE_MOMENTS_KERNEL_LAUNCHER(int64_t, float);
DEFINE_MOMENTS_KERNEL_LAUNCHER(float, float);
DEFINE_MOMENTS_KERNEL_LAUNCHER(double, double);

template <> void Moments<float16, float, CUDAContext>(
    const int               ndims,
    const int*              dims,
    const int               naxes,
    const int*              axes,
    const float16*          x,
    float*                  mean,
    float*                  var,
    CUDAContext*            ctx) {
    _Moments(
        ndims, dims,
        naxes, axes,
        reinterpret_cast<const half*>(x),
        mean, var, ctx
    );
}

#undef FIXED_DIVISOR_DIV_MOD
#undef DEFINE_MOMENTS_KERNEL_LAUNCHER

}  // namespace kernel

}  // namespace dragon

#endif  // WITH_CUDA