
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/cast.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/* <T = ?, Device = CUDA> */

template <typename T>
__global__ void _ArgMax(
    const int               nthreads,
    const int               inner_dim,
    const int               axis_dim,
    const T*                x,
    int64_t*                indices,
    T*                      values) {
    CUDA_1D_KERNEL_LOOP(yi, nthreads) {
        const int oix = yi / inner_dim;
        const int iix = yi % inner_dim;
        const T* X = x + (oix * axis_dim * inner_dim + iix);
        T max_val = X[0], val; int64_t max_idx = 0;
        for (int j = 1; j < axis_dim; ++j) {
            val = X[j * inner_dim];
            if (val > max_val) {
                max_val = val;
                max_idx = j;
            }
        }
        indices[yi] = max_idx;
        if (values) values[yi] = max_val;
    }
}

/* <T = float16, Device = CUDA> */

template<> __global__ void _ArgMax<half>(
    const int               nthreads,
    const int               inner_dim,
    const int               axis_dim,
    const half*             x,
    int64_t*                indices,
    half*                   values) {
    CUDA_1D_KERNEL_LOOP(yi, nthreads) {
#if __CUDA_ARCH__ >= 530
        const int oix = yi / inner_dim;
        const int iix = yi % inner_dim;
        const half* X = x + (oix * axis_dim * inner_dim + iix);
        half max_val = X[0], val; int64_t max_idx = 0;
        for (int j = 1; j < axis_dim; ++j) {
            val = X[j * inner_dim];
            if (__hgt(val, max_val)) {
                max_val = val;
                max_idx = j;
            }
        }
        indices[yi] = max_idx;
        if (values) values[yi] = max_val;
#endif
    }
}

/* <T = ?, Device = CUDA> */

template <typename T>
__global__ void _ArgMin(
    const int               nthreads,
    const int               inner_dim,
    const int               axis_dim,
    const T*                x,
    int64_t*                indices,
    T*                      values) {
    CUDA_1D_KERNEL_LOOP(yi, nthreads) {
        const int oix = yi / inner_dim;
        const int iix = yi % inner_dim;
        const T* X = x + (oix * axis_dim * inner_dim + iix);
        T min_val = X[0], val; int64_t min_idx = 0;
        for (int j = 1; j < axis_dim; ++j) {
            val = X[j * inner_dim];
            if (val < min_val) {
                min_val = val;
                min_idx = j;
            }
        }
        indices[yi] = min_idx;
        if (values) values[yi] = min_val;
    }
}

/* <T = float16, Device = CUDA> */

template<> __global__ void _ArgMin<half>(
    const int               nthreads,
    const int               inner_dim,
    const int               axis_dim,
    const half*             x,
    int64_t*                indices,
    half*                   values) {
    CUDA_1D_KERNEL_LOOP(yi, nthreads) {
#if __CUDA_ARCH__ >= 530
        const int oix = yi / inner_dim;
        const int iix = yi % inner_dim;
        const half* X = x + (oix * axis_dim * inner_dim + iix);
        half max_val = X[0], val; int64_t max_idx = 0;
        for (int j = 1; j < axis_dim; ++j) {
            val = X[j * inner_dim];
            if (__hlt(val, max_val)) {
                max_val = val;
                max_idx = j;
            }
        }
        indices[yi] = max_idx;
        if (values) values[yi] = max_val;
#endif
    }
}

/* Kernel Launchers */

#define DEFINE_ARGREDUCE_KERNEL_LAUNCHER(name, T) \
    template<> void name<T, CUDAContext>( \
        const int               outer_dim, \
        const int               inner_dim, \
        const int               axis_dim, \
        const int               top_k, \
        const T*                x, \
        int64_t*                indices, \
        T*                      values, \
        CUDAContext*            ctx) { \
        CHECK_EQ(top_k, 1) << "\nRequired top_k == 1."; \
        auto nthreads = outer_dim * inner_dim; \
        _##name \
            <<< CUDA_BLOCKS(nthreads), CUDA_THREADS, \
                0, ctx->cuda_stream() >>>( \
            nthreads, inner_dim, axis_dim, \
            x, indices, values \
        ); \
    }

DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMax, bool);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMax, int8_t);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMax, uint8_t);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMax, int);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMax, int64_t);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMax, float);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMax, double);

DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMin, bool);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMin, int8_t);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMin, uint8_t);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMin, int);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMin, int64_t);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMin, float);
DEFINE_ARGREDUCE_KERNEL_LAUNCHER(ArgMin, double);

template<> void ArgMax<float16, CUDAContext>(
    const int               outer_dim,
    const int               inner_dim,
    const int               axis_dim,
    const int               top_k,
    const float16*          x,
    int64_t*                indices,
    float16*                values,
    CUDAContext*            ctx) {
    CHECK_EQ(top_k, 1) << "\nRequired top_k == 1.";
    auto nthreads = outer_dim * inner_dim;
    _ArgMax
        <<< CUDA_BLOCKS(nthreads), CUDA_THREADS,
            0, ctx->cuda_stream() >>>(
        nthreads, inner_dim, axis_dim,
        reinterpret_cast<const half*>(x),
        indices,
        reinterpret_cast<half*>(values)
    );
}

template<> void ArgMin<float16, CUDAContext>(
    const int               outer_dim,
    const int               inner_dim,
    const int               axis_dim,
    const int               top_k,
    const float16*          x,
    int64_t*                indices,
    float16*                values,
    CUDAContext*            ctx) {
    CHECK_EQ(top_k, 1) << "\nRequired top_k == 1.";
    auto nthreads = outer_dim * inner_dim;
    _ArgMin
        <<< CUDA_BLOCKS(nthreads), CUDA_THREADS,
            0, ctx->cuda_stream() >>>(
         nthreads, inner_dim, axis_dim,
         reinterpret_cast<const half*>(x),
         indices,
         reinterpret_cast<half*>(values)
    );
}

#undef DEFINE_ARGREDUCE_KERNEL_LAUNCHER

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA