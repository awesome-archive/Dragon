
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

#define FIXED_DIVISOR_DIV_MOD(d, n, q, r) \
    do {                                  \
        const auto n_copy = n;            \
        *q = n_copy / d;                  \
        *r = n_copy % d;                  \
    } while (0)

/* <T = ?, Device = CUDA> */

template<typename T>
__global__ void _Crop(
    const int               nthreads,
    const int               ndims,
    const int*              x_strides,
    const int*              y_dims,
    const int*              starts,
    const T*                x,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(yi, nthreads) {
        int xi = 0, tmp = yi;
#pragma unroll
        for (int d = ndims - 1; d >= 0; --d) {
            int r;
#if __CUDA_ARCH__ >= 350
            FIXED_DIVISOR_DIV_MOD(__ldg(y_dims + d), tmp, &tmp, &r);
            xi += (r + __ldg(starts + d)) * __ldg(x_strides + d);
#else
            FIXED_DIVISOR_DIV_MOD(y_dims[d], tmp, &tmp, &r);
            xi += (r + starts[d]) * x_strides[d];
#endif
        }
        y[yi] = x[xi];
    }
}

/* <T = ?, Device = CUDA> */

template<typename T>
__global__ void _CropGrad(
    const int               nthreads,
    const int               ndims,
    const int*              x_strides,
    const int*              y_dims,
    const int*              starts,
    const T*                dy,
    T*                      dx) {
    CUDA_1D_KERNEL_LOOP(yi, nthreads) {
        int xi = 0, tmp = yi;
#pragma unroll
        for (int d = ndims - 1; d >= 0; --d) {
            int r;
#if __CUDA_ARCH__ >= 350
            FIXED_DIVISOR_DIV_MOD(__ldg(y_dims + d), tmp, &tmp, &r);
            xi += (r + __ldg(starts + d)) * __ldg(x_strides + d);
#else
            FIXED_DIVISOR_DIV_MOD(y_dims[d], tmp, &tmp, &r);
            xi += (r + starts[d]) * x_strides[d];
#endif
        }
        dx[xi] = dy[yi];
    }
}

/* Kernel Launchers */

#define DEFINE_CROP_KERNEL_LAUNCHER(name, T) \
    template<> void name<T, CUDAContext>( \
        const int               count, \
        const int               ndims, \
        const int*              x_strides, \
        const int*              y_dims, \
        const int*              starts, \
        const T*                x, \
        T*                      y, \
        CUDAContext*            ctx) { \
        _##name \
            << < CUDA_BLOCKS(count), CUDA_THREADS, \
                 0, ctx->cuda_stream() >> >( \
            count, ndims, \
            x_strides, y_dims, \
            starts, x, y \
        ); \
    }

DEFINE_CROP_KERNEL_LAUNCHER(Crop, bool);
DEFINE_CROP_KERNEL_LAUNCHER(Crop, int8_t);
DEFINE_CROP_KERNEL_LAUNCHER(Crop, uint8_t);
DEFINE_CROP_KERNEL_LAUNCHER(Crop, int);
DEFINE_CROP_KERNEL_LAUNCHER(Crop, int64_t);
DEFINE_CROP_KERNEL_LAUNCHER(Crop, float16);
DEFINE_CROP_KERNEL_LAUNCHER(Crop, float);
DEFINE_CROP_KERNEL_LAUNCHER(Crop, double);

DEFINE_CROP_KERNEL_LAUNCHER(CropGrad, bool);
DEFINE_CROP_KERNEL_LAUNCHER(CropGrad, int8_t);
DEFINE_CROP_KERNEL_LAUNCHER(CropGrad, uint8_t);
DEFINE_CROP_KERNEL_LAUNCHER(CropGrad, int);
DEFINE_CROP_KERNEL_LAUNCHER(CropGrad, int64_t);
DEFINE_CROP_KERNEL_LAUNCHER(CropGrad, float16);
DEFINE_CROP_KERNEL_LAUNCHER(CropGrad, float);
DEFINE_CROP_KERNEL_LAUNCHER(CropGrad, double);

#undef FIXED_DIVISOR_DIV_MOD
#undef DEFINE_CROP_KERNEL_LAUNCHER

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA