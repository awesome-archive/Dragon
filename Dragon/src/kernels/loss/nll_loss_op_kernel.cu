
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/*! <Tx = float32, Ty = ?, Device = CUDA> */

template <typename Tx, typename Ty>
__global__ void _NLLLoss(
    const int               nthreads,
    const int               axis_dim,
    const int               inner_dim,
    const int               nignores,
    const int*              ignore,
    const Tx*               log_prob,
    const Ty*               target,
    Tx*                     loss,
    int*                    flag) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        const int oix = i / inner_dim;
        const int iix = i % inner_dim;
        const int label = target[oix * inner_dim + iix];
        int k;
        for (k = 0; k < nignores; k++) {
            if (label == ignore[k]) {
                loss[i] = flag[i] = 0;
                break;
            }
        }
        if (k == nignores) {
            loss[i] = -log_prob[
                (oix * axis_dim + label
                   ) * inner_dim + iix];
            flag[i] = 1;
        }
    }
}

/*! <Tx = float32, Ty = float32, Device = CUDA> */

template <> void NLLLoss<float, float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const int               nignores,
    const int*              ignore,
    const float*            log_prob,
    const float*            target,
    float*                  loss,
    int*                    flag,
    CUDAContext*            ctx) {
    auto nthreads = outer_dim * inner_dim;
    _NLLLoss
        << < CUDA_BLOCKS(nthreads), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        nthreads, axis_dim, inner_dim, nignores,
        ignore, log_prob, target, loss, flag
     );
}

/*! <Tx = float32, Ty = int64, Device = CUDA> */

template <> void NLLLoss<float, int64_t, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const int               nignores,
    const int*              ignore,
    const float*            log_prob,
    const int64_t*          target,
    float*                  loss,
    int*                    flag,
    CUDAContext*            ctx) {
    auto nthreads = outer_dim * inner_dim;
    _NLLLoss
        << < CUDA_BLOCKS(nthreads), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        nthreads, axis_dim, inner_dim, nignores,
        ignore, log_prob, target, loss, flag
    );
}

/*! <Tx = ?, Ty = ?, Device = CUDA> */

template <typename Tx, typename Ty>
__global__ void _NLLLossGrad(
    const int               nthreads,
    const int               axis_dim,
    const int               inner_dim,
    const int               nignores,
    const int*              ignore,
    const Tx*               log_prob,
    const Ty*               target,
    Tx*                     dx,
    int*                    flag) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        const int oix = i / inner_dim;
        const int iix = i % inner_dim;
        const int label = target[oix * inner_dim + iix];
        int k;
        for (k = 0; k < nignores; k++)
            if (label == ignore[k]) break;
        if (k != nignores) {
            flag[i] = 0;
        } else {
            dx[(oix * axis_dim + label
                  ) * inner_dim + iix] = -1;
            flag[i] = 1;
        }
    }
}

/*! <Tx = float32, Ty = float32, Device = CUDA> */

template<> void NLLLossGrad<float, float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const int               nignores,
    const int*              ignore,
    const float*            log_prob,
    const float*            target,
    float*                  dx,
    int*                    flag,
    CUDAContext*            ctx) {
    auto nthreads = outer_dim * inner_dim;
    _NLLLossGrad
        << < CUDA_BLOCKS(nthreads), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        nthreads, axis_dim, inner_dim, nignores,
        ignore, log_prob, target, dx, flag
    );
}

/*! <Tx = float32, Ty = int64, Device = CUDA> */

template<> void NLLLossGrad<float, int64_t, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const int               nignores,
    const int*              ignore,
    const float*            log_prob,
    const int64_t*          target,
    float*                  dx,
    int*                    flag,
    CUDAContext*            ctx) {
    auto nthreads = outer_dim * inner_dim;
    _NLLLossGrad
        << < CUDA_BLOCKS(nthreads), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        nthreads, axis_dim, inner_dim, nignores,
        ignore, log_prob, target, dx, flag
    );
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA