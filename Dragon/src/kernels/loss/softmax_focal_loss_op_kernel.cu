
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/*! SoftmaxFocalLoss <Tx = ?, Ty = ?, Device = CUDA> */

template <typename Tx, typename Ty>
__global__ void _SoftmaxFocalLoss(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const int               num_ignores,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const Tx*               prob,
    const Ty*               labels,
    const int*              ignores,
    Tx*                     losses,
    int*                    flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        const int label = labels[oix * inner_dim + iix];
        int k;
        for (k = 0; k < num_ignores; k++) {
            if (label == ignores[k]) {
                losses[idx] = flags[idx] = 0;
                break;
            }
        }
        if (k == num_ignores) {
            const int t = (oix * axis_dim + label) * inner_dim + iix;
            Tx scale = pow(1 - prob[t], gamma);
            scale = label > neg_id ?
                pos_alpha * scale : neg_alpha * scale;
            losses[idx] = -scale * log(max(prob[t], FLT_MIN));
            flags[idx] = label > neg_id ? 1 : 0;
        }
    }
}

/*! SoftmaxFocalLoss <Tx = float32, Ty = float32, Device = CUDA> */

template <> void SoftmaxFocalLoss<float, float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const int               num_ignores,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const float*            prob,
    const float*            labels,
    const int*              ignores,
    float*                  losses,
    int*                    flags,
    CUDAContext*            ctx) {
    const auto num_preds = outer_dim * inner_dim;
    _SoftmaxFocalLoss<float, float>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (num_preds, axis_dim, inner_dim, num_ignores,
            pos_alpha, neg_alpha, gamma, neg_id,
                prob, labels, ignores, losses, flags);
}

/*! SoftmaxFocalLoss <Tx = float32, Ty = int64, Device = CUDA> */

template <> void SoftmaxFocalLoss<float, int64_t, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const int               num_ignores,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const float*            prob,
    const int64_t*          labels,
    const int*              ignores,
    float*                  losses,
    int*                    flags,
    CUDAContext*            ctx) {
    const auto num_preds = outer_dim * inner_dim;
    _SoftmaxFocalLoss<float, int64_t>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (num_preds, axis_dim, inner_dim, num_ignores,
            pos_alpha, neg_alpha, gamma, neg_id,
                prob, labels, ignores, losses, flags);
}

/*! SoftmaxFocalLossGrad <Tx = ?, Ty = ?, Device = CUDA> */

template <typename Tx, typename Ty>
__global__ void _SoftmaxFocalLossGrad(
    const int               count,
    const int               axis_dim,
    const int               inner_dim,
    const int               num_ignores,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const Tx*               prob,
    const Ty*               labels,
    const int*              ignores,
    Tx*                     dx,
    int*                    flags) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
        const int oix = idx / inner_dim;
        const int iix = idx % inner_dim;
        const int label = labels[oix * inner_dim + iix];
        int k;
        for (k = 0; k < num_ignores; k++)
            if (label == ignores[k]) break;
        if (k != num_ignores) {
            for (int c = 0; c < axis_dim; c++)
                dx[(oix * axis_dim + c) * inner_dim + iix] = (Tx)0;
            flags[idx] = 0;
        } else {
            const int t = (oix * axis_dim + label) * inner_dim + iix;
            Tx onemp = 1 - prob[t];
            // Unstable if gamma is 0
            Tx grad = -gamma * pow(onemp, gamma - 1)
                             * log(max(prob[t], FLT_MIN))
                             * prob[t] + pow(onemp, gamma);
            grad = label > neg_id ?
                pos_alpha * grad : neg_alpha * grad;
            for (int c = 0; c < axis_dim; c++) {
                const int i = (oix * axis_dim + c) * inner_dim + iix;
                if (c == label) {
                    dx[i] = grad * (prob[t] - 1);
                } else {
                    dx[i] = grad * prob[i];
                }
            }
            flags[idx] = label > neg_id ? 1 : 0;
        }
    }
}

/*! SoftmaxFocalLossGrad <Tx = float32, Ty = float32, Device = CUDA> */

template<> void SoftmaxFocalLossGrad<float, float, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const int               num_ignores,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const float*            prob,
    const float*            labels,
    const int*              ignores,
    float*                  dx,
    int*                    flags,
    CUDAContext*            ctx) {
    const int num_preds = outer_dim * inner_dim;
    _SoftmaxFocalLossGrad<float, float>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (num_preds, axis_dim, inner_dim, num_ignores,
            pos_alpha, neg_alpha, gamma, neg_id,
                prob, labels, ignores, dx, flags);
}

/*! SoftmaxFocalLossGrad <Tx = float32, Ty = int64, Device = CUDA> */

template<> void SoftmaxFocalLossGrad<float, int64_t, CUDAContext>(
    const int               outer_dim,
    const int               axis_dim,
    const int               inner_dim,
    const int               num_ignores,
    const float             pos_alpha,
    const float             neg_alpha,
    const float             gamma,
    const int               neg_id,
    const float*            prob,
    const int64_t*          labels,
    const int*              ignores,
    float*                  dx,
    int*                    flags,
    CUDAContext*            ctx) {
    const int num_preds = outer_dim * inner_dim;
    _SoftmaxFocalLossGrad<float, int64_t>
        << < CUDA_BLOCKS(num_preds), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (num_preds, axis_dim, inner_dim, num_ignores,
            pos_alpha, neg_alpha, gamma, neg_id,
                prob, labels, ignores, dx, flags);
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA