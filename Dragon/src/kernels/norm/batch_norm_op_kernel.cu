
#include <hip/hip_runtime.h>
/*!
 * Codes are based on:
 *
 *    <https://github.com/pytorch/pytorch/blob/master/caffe2/operators/group_norm_op.cu>
 *
 * ------------------------------------------------------------
 */

#ifdef WITH_CUDA

#include "core/mixedmem.h"
#include "utils/op_kernel.h"
#include "utils/math_utils.h"
#include "utils/cub_device.h"

namespace dragon {

namespace kernel {

/*! BatchNormBackwardTraining <T = ?, Device = CUDA> */

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormInternalGrad(
    const int                   N,
    const int                   C,
    const int                   S,
    const Tx*                   x,
    const Tp*                   mu,
    const Tp*                   rsig,
    const Tp*                   gamma,
    const Tx*                   dy,
    Tp*                         ds,
    Tp*                         db,
    Tp*                         dgamma,
    Tp*                         dbeta) {
    const int outer_dim = N * S;
    __shared__ typename BlockReduce<Tp>::TempStorage ds_storage;
    __shared__ typename BlockReduce<Tp>::TempStorage db_storage;
    __shared__ typename BlockReduce<Tp>::TempStorage dga_storage;
    __shared__ typename BlockReduce<Tp>::TempStorage dbe_storage;
    CUDA_2D_KERNEL_LOOP1(i, C) {
        Tp ds_val = 0, db_val = 0;
        Tp dga_val = 0, dbe_val = 0;
        CUDA_2D_KERNEL_LOOP2(j, outer_dim) {
            const int idx = kOrder == StorageOrder::NCHW ?
                (j / S * C + i) * S + j % S : j * C + i;
#if __CUDA_ARCH__ >= 350
            ds_val += __ldg(gamma + i) * __ldg(dy + idx) * __ldg(x + idx);
            db_val += __ldg(gamma + i) * __ldg(dy + idx);
            dga_val += __ldg(dy + idx) *(
                __ldg(x + idx) - __ldg(mu + i)
            ) * __ldg(rsig + i);
            dbe_val += __ldg(dy + idx);
#else
            ds_val += gamma[i] * dy[idx] * x[idx];
            db_val += gamma[i] * dy[idx];
            dga_val += dy[idx] * (x[idx] - mu[i]) * rsig[i];
            dbe_val += dy[idx];
#endif
        }
        ds_val = BlockReduce<Tp>(ds_storage).Reduce(ds_val, cub::Sum());
        db_val = BlockReduce<Tp>(db_storage).Reduce(db_val, cub::Sum());
        dga_val = BlockReduce<Tp>(dga_storage).Reduce(dga_val, cub::Sum());
        dbe_val = BlockReduce<Tp>(dbe_storage).Reduce(dbe_val, cub::Sum());
        if (threadIdx.x == 0) {
            ds[i] = ds_val; db[i] = db_val;
            // Accumulate the gradients of trainable parameters
            dgamma[i] += dga_val; dbeta[i] += dbe_val;
        }
    }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormTrainingGrad(
    const int                   nthreads,
    const int                   N,
    const int                   C,
    const int                   S,
    const Tx*                   x,
    const Tp*                   mu,
    const Tp*                   rsig,
    const Tp*                   gamma,
    const Tp*                   ds,
    const Tp*                   db,
    const Tx*                   dy,
    Tx*                         dx) {
    const Tp denom = Tp(1) / static_cast<Tp>(N * S);
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        const int i_param = kOrder == StorageOrder::NCHW ?
            (i / S) % C : i % C;
#if __CUDA_ARCH__ >= 350
        const Tp u = (
            __ldg(db + i_param) * __ldg(mu + i_param) - __ldg(ds + i_param)
        ) * (__ldg(x + i) - __ldg(mu + i_param)
        ) * utils::math::Cube<Tp>(__ldg(rsig + i_param));
        const Tp v = __ldg(db + i_param) * __ldg(rsig + i_param);
        dx[i] = __ldg(gamma + i_param) * __ldg(dy + i) *
            __ldg(rsig + i_param) + (u - v) * denom;
#else
        const Tp u = (db[i_param] * mu[i_param] - ds[i_param]) *
            (x[i] - mu[i_param]) * utils::math::Cube<Tp>(rsig[i_param]);
        const Tp v = db[i_param] * rsig[i_param];
        dx[i] = gamma[i_param] * dy[i] * rsig[i_param] + (u - v) * denom;
#endif
    }
}

/*! BatchNormBackwardInference <T = ?, Device = CUDA> */

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormWGrad(
    const int                   N,
    const int                   C,
    const int                   S,
    const Tx*                   x,
    const Tp*                   mu,
    const Tp*                   rsig,
    const Tx*                   dy,
    Tp*                         dgamma,
    Tp*                         dbeta) {
    const int outer_dim = N * S;
    __shared__ typename BlockReduce<Tp>::TempStorage dg_storage;
    __shared__ typename BlockReduce<Tp>::TempStorage db_storage;
    CUDA_2D_KERNEL_LOOP1(i, C) {
        Tp dg_val = 0, db_val = 0;
        CUDA_2D_KERNEL_LOOP2(j, outer_dim) {
            const int idx = kOrder == StorageOrder::NCHW ?
                (j / S * C + i) * S + j % S : j * C + i;
#if __CUDA_ARCH__ >= 350
            dg_val += __ldg(dy + idx) * (
                __ldg(x + idx) - __ldg(mu + i)
            ) * __ldg(rsig + i);
            db_val += __ldg(dy + idx);
#else
            dg_val += dy[idx] * (x[idx] - mu[i]) * rsig[i];
            db_val += dy[idx];
#endif
        }
        dg_val = BlockReduce<Tp>(dg_storage).Reduce(dg_val, cub::Sum());
        db_val = BlockReduce<Tp>(db_storage).Reduce(db_val, cub::Sum());
        if (threadIdx.x == 0) {
            // Accumulate the gradients of trainable parameters
            dgamma[i] += dg_val; dbeta[i] += db_val;
        }
    }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormInferenceGrad(
    const int                   nthreads,
    const int                   C,
    const int                   S,
    const Tp*                   rsig,
    const Tp*                   gamma,
    const Tx*                   dy,
    Tx*                         dx) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        const int i_param = kOrder == StorageOrder::NCHW ?
            (i / S) % C : i % C;
#if __CUDA_ARCH__ >= 350
        dx[i] = __ldg(gamma + i_param) * __ldg(dy + i)
                       * __ldg(rsig + i_param);
#else
        dx[i] = gamma[i_param] * dy[i] * rsig[i_param];
#endif
    }
}

/*! Kernel Launchers */

#define DEFINE_BACKWARD_KERNEL_LAUNCHER(Tx, Tp) \
    template <> void BatchNormBackwardTraining<Tx, Tp, CUDAContext>( \
        const int                   N, \
        const int                   C, \
        const int                   S, \
        const string&               data_format, \
        const Tx*                   x, \
        const Tp*                   mu, \
        const Tp*                   rsig, \
        const Tp*                   gamma, \
        const Tx*                   dy, \
        Tp*                         ds, \
        Tp*                         db, \
        Tx*                         dx, \
        Tp*                         dgamma, \
        Tp*                         dbeta, \
        CUDAContext*                ctx) { \
        auto nthreads = N * C * S; \
        if (data_format == "NCHW") { \
            _BatchNormInternalGrad<Tx, Tp, StorageOrder::NCHW> \
                << < CUDA_2D_BLOCKS(C), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (N, C, S, x, mu, rsig, gamma, dy, \
                    ds, db, dgamma, dbeta); \
            _BatchNormTrainingGrad<Tx, Tp, StorageOrder::NCHW> \
                << < CUDA_BLOCKS(nthreads), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (nthreads, N, C, S, x, mu, rsig, gamma, ds, db, dy, dx); \
        } else if (data_format == "NHWC") { \
            _BatchNormInternalGrad<Tx, Tp, StorageOrder::NHWC> \
                << < CUDA_2D_BLOCKS(C), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (N, C, S, x, mu, rsig, gamma, dy, \
                    ds, db, dgamma, dbeta); \
            _BatchNormTrainingGrad<Tx, Tp, StorageOrder::NHWC> \
                << < CUDA_BLOCKS(nthreads), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (nthreads, N, C, S, x, mu, rsig, gamma, ds, db, dy, dx); \
        } \
    } \
    template <> void BatchNormBackwardInference<Tx, Tp, CUDAContext>( \
        const int                   N, \
        const int                   C, \
        const int                   S, \
        const string&               data_format, \
        const Tx*                   x, \
        const Tp*                   mu, \
        const Tp*                   rsig, \
        const Tp*                   gamma, \
        const Tx*                   dy, \
        Tx*                         dx, \
        Tp*                         dgamma, \
        Tp*                         dbeta, \
        CUDAContext*                ctx) { \
        auto nthreads = N * C * S; \
        if (data_format == "NCHW") { \
            if (dgamma != nullptr) { \
                _BatchNormWGrad<Tx, Tp, StorageOrder::NCHW> \
                    << < CUDA_2D_BLOCKS(C), CUDA_THREADS, \
                         0, ctx->cuda_stream() >> > \
                    (N, C, S, x, mu, rsig, dy, dgamma, dbeta); \
            } \
            _BatchNormInferenceGrad<Tx, Tp, StorageOrder::NCHW> \
                << < CUDA_BLOCKS(nthreads), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (nthreads, C, S, rsig, gamma, dy, dx); \
        } else if (data_format == "NHWC") { \
            if (dgamma != nullptr) { \
                _BatchNormWGrad<Tx, Tp, StorageOrder::NHWC> \
                    << < CUDA_2D_BLOCKS(C), CUDA_THREADS, \
                         0, ctx->cuda_stream() >> > \
                    (N, C, S, x, mu, rsig, dy, dgamma, dbeta); \
            } \
            _BatchNormInferenceGrad<Tx, Tp, StorageOrder::NHWC> \
                << < CUDA_BLOCKS(nthreads), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (nthreads, C, S, rsig, gamma, dy, dx); \
        } \
    }

DEFINE_BACKWARD_KERNEL_LAUNCHER(float, float);
#undef DEFINE_BACKWARD_KERNEL_LAUNCHER

}  // namespace kernel

}  // namespace dragon

#endif  // WITH_CUDA