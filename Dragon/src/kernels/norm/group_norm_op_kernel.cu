
#include <hip/hip_runtime.h>
/*!
 * Codes are based on:
 *
 *    <https://github.com/pytorch/pytorch/blob/master/caffe2/operators/group_norm_op.cu>
 *
 * ------------------------------------------------------------
 */

#ifdef WITH_CUDA

#include "core/mixedmem.h"
#include "utils/op_kernel.h"
#include "utils/math_utils.h"
#include "utils/cub_device.h"

namespace dragon {

namespace kernel {

#define L(x, i) __ldg(x + i)
#define LF(x, i) __half2float(__ldg(x + i))

/*! GroupNormForward <T = ?, Device = CUDA> */

template <typename T>
__global__ void _GroupNormFusedParams(
    const int                   N,
    const int                   G,
    const int                   D,
    const T*                    mu,
    const T*                    rsig,
    const T*                    gamma,
    const T*                    beta,
    T*                          scale,
    T*                          bias) {
    const int outer_dim = N * G;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        const int g = i % G;
#if __CUDA_ARCH__ >= 350
        const T mu_val = L(mu, i);
        const T rsig_val = L(rsig, i);
#else
        const T mu_val = mu[i];
        const T rsig_val = rsig[i];
#endif
        CUDA_2D_KERNEL_LOOP2(j, D) {
            const int i_scale = i * D + j;
            const int i_gamma = g * D + j;
#if __CUDA_ARCH__ >= 350
            const T scale_val = L(gamma, i_gamma) * rsig_val;
            scale[i_scale] = scale_val;
            bias[i_scale] = L(beta, i_gamma) - scale_val * mu_val;
#else
            const T scale_val = gamma[i_gamma] * rsig_val;
            scale[i_scale] = scale_val;
            bias[i_scale] = beta[i_gamma] - scale_val * mu_val;
#endif
        }
    }
}

template <typename Tx, typename Tp>
__global__ void _GroupNormForwardNCHW(
    const int                   N,
    const int                   C,
    const int                   S,
    const Tx*                   x,
    const Tp*                   scale,
    const Tp*                   bias,
    Tx*                         y) {
    const int outer_dim = N * C;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
#if __CUDA_ARCH__ >= 350
        const Tp scale_val = L(scale, i);
        const Tp bias_val = L(bias, i);
#else
        const Tp scale_val = scale[i];
        const Tp bias_val = bias[i];
#endif
        CUDA_2D_KERNEL_LOOP2(j, S) {
            const int idx = i * S + j;
#if __CUDA_ARCH__ >= 350
            y[idx] = L(x, idx) * scale_val + bias_val;
#else
            y[idx] = x[idx] * scale_val + bias_val;
#endif
        }
    }
}

template<> __global__ void _GroupNormForwardNCHW<half, float>(
    const int                   N,
    const int                   C,
    const int                   S,
    const half*                 x,
    const float*                scale,
    const float*                bias,
    half*                       y) {
#if __CUDA_ARCH__ >= 530
    const int outer_dim = N * C;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        const float scale_val = L(scale, i);
        const float bias_val = L(bias, i);
        CUDA_2D_KERNEL_LOOP2(j, S) {
            const int idx = i * S + j;
            y[idx] = __float2half(
                LF(x, idx) * scale_val + bias_val
            );
        }
    }
#endif
}

template <typename Tx, typename Tp>
__global__ void _GroupNormForwardNHWC(
    const int                   N,
    const int                   C,
    const int                   S,
    const Tx*                   x,
    const Tp*                   scale,
    const Tp*                   bias,
    Tx*                         y) {
    const int outer_dim = N * S;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        const int n = i / S;
        CUDA_2D_KERNEL_LOOP2(j, C) {
            const int idx = i * C + j;
            const int i_scale = n * C + j;
#if __CUDA_ARCH__ >= 350
            y[idx] = L(x, idx) * L(scale, i_scale)
                         + L(bias, i_scale);
#else
            y[idx] = x[idx] * scale[i_scale] + bias[i_scale];
#endif
        }
    }
}

template<> __global__ void _GroupNormForwardNHWC<half, float>(
    const int                   N,
    const int                   C,
    const int                   S,
    const half*                 x,
    const float*                scale,
    const float*                bias,
    half*                       y) {
#if __CUDA_ARCH__ >= 530
    const int outer_dim = N * S;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        const int n = i / S;
        CUDA_2D_KERNEL_LOOP2(j, C) {
            const int idx = i * C + j;
            const int i_scale = n * C + j;
            y[idx] = __float2half(
                LF(x, idx) * L(scale, i_scale)
                    + L(bias, i_scale)
            );
        }
    }
#endif
}

/*! GroupNormBackward <T = ?, Device = CUDA> */

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _GroupNormWGrad(
    const int                   N,
    const int                   G,
    const int                   D,
    const int                   S,
    const Tx*                   x,
    const Tp*                   mu,
    const Tp*                   rsig,
    const Tx*                   dy,
    Tp*                         dgamma,
    Tp*                         dbeta) {
    const int outer_dim = G * D;
    const int inner_dim = N * S;
    __shared__ typename BlockReduce<Tp>::TempStorage dg_storage;
    __shared__ typename BlockReduce<Tp>::TempStorage db_storage;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        Tp dg_val = 0, db_val = 0;
        CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
            const int n = j / S;
            const int idx = kOrder == StorageOrder::NCHW ?
                (n * outer_dim + i) * S + j % S :
                 j * outer_dim + i;
            const int i_mu = n * G + i / D;
#if __CUDA_ARCH__ >= 350
            dg_val += L(dy, idx) * (
                L(x, idx) - L(mu, i_mu)
            ) * L(rsig, i_mu);
            db_val += L(dy, idx);
#else
            dg_val += dy[idx] * (x[idx] - mu[i_mu]) * rsig[i_mu];
            db_val += dy[idx];
#endif
        }
        dg_val = BlockReduce<Tp>(dg_storage).Reduce(dg_val, cub::Sum());
        db_val = BlockReduce<Tp>(db_storage).Reduce(db_val, cub::Sum());
        if (threadIdx.x == 0) {
            dgamma[i] = dg_val; dbeta[i] = db_val;
        }
    }
}

template <StorageOrder kOrder>
__global__ void _GroupNormWGradHalf(
    const int                   N,
    const int                   G,
    const int                   D,
    const int                   S,
    const half*                 x,
    const float*                mu,
    const float*                rsig,
    const half*                 dy,
    float*                      dgamma,
    float*                      dbeta) {
#if __CUDA_ARCH__ >= 530
    const int outer_dim = G * D;
    const int inner_dim = N * S;
    __shared__ typename BlockReduce<float>::TempStorage dg_storage;
    __shared__ typename BlockReduce<float>::TempStorage db_storage;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        float dg_val = 0.f, db_val = 0.f;
        CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
            const int n = j / S;
            const int idx = kOrder == StorageOrder::NCHW ?
                (n * outer_dim + i) * S + j % S :
                 j * outer_dim + i;
            const int i_mu = n * G + i / D;
            dg_val += LF(dy, idx) * (
                LF(x, idx) - L(mu, i_mu)
            ) * L(rsig, i_mu);
            db_val += LF(dy, idx);
        }
        dg_val = BlockReduce<float>(dg_storage).Reduce(dg_val, cub::Sum());
        db_val = BlockReduce<float>(db_storage).Reduce(db_val, cub::Sum());
        if (threadIdx.x == 0) {
            dgamma[i] = dg_val; dbeta[i] = db_val;
        }
    }
#endif
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _GroupNormInternalGrad(
    const int                   N,
    const int                   G,
    const int                   D,
    const int                   S,
    const Tx*                   x,
    const Tp*                   gamma,
    const Tx*                   dy,
    Tp*                         ds,
    Tp*                         db) {
    const int outer_dim = N * G;
    const int inner_dim = D * S;
    __shared__ typename BlockReduce<Tp>::TempStorage ds_storage;
    __shared__ typename BlockReduce<Tp>::TempStorage db_storage;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        Tp ds_val = 0, db_val = 0;
        CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
            const int i_gamma = i % G * D + j / S;
            const int idx = kOrder == StorageOrder::NCHW
                ? i * inner_dim + j :
                  (i / G * S + j % S) * G * D + i_gamma;
#if __CUDA_ARCH__ >= 350
            ds_val += L(gamma, i_gamma) * L(dy, idx) * L(x, idx);
            db_val += L(gamma, i_gamma) * L(dy, idx);
#else
            ds_val += gamma[i_gamma] * dy[idx] * x[idx];
            db_val += gamma[i_gamma] * dy[idx];
#endif
        }
        ds_val = BlockReduce<Tp>(ds_storage).Reduce(ds_val, cub::Sum());
        db_val = BlockReduce<Tp>(db_storage).Reduce(db_val, cub::Sum());
        if (threadIdx.x == 0) {
            ds[i] = ds_val; db[i] = db_val;
        }
    }
}

template <StorageOrder kOrder>
__global__ void _GroupNormInternalGradHalf(
    const int                   N,
    const int                   G,
    const int                   D,
    const int                   S,
    const half*                 x,
    const float*                gamma,
    const half*                 dy,
    float*                      ds,
    float*                      db) {
#if __CUDA_ARCH__ >= 530
    const int outer_dim = N * G;
    const int inner_dim = D * S;
    __shared__ typename BlockReduce<float>::TempStorage ds_storage;
    __shared__ typename BlockReduce<float>::TempStorage db_storage;
    CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
        float ds_val = 0.f, db_val = 0.f;
        CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
            const int i_gamma = i % G * D + j / S;
            const int idx = kOrder == StorageOrder::NCHW
                ? i * inner_dim + j :
                  (i / G * S + j % S) * G * D + i_gamma;
            ds_val += L(gamma, i_gamma) * LF(dy, idx) * LF(x, idx);
            db_val += L(gamma, i_gamma) * LF(dy, idx);
        }
        ds_val = BlockReduce<float>(ds_storage).Reduce(ds_val, cub::Sum());
        db_val = BlockReduce<float>(db_storage).Reduce(db_val, cub::Sum());
        if (threadIdx.x == 0) {
            ds[i] = ds_val; db[i] = db_val;
        }
    }
#endif
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _GroupNormGrad(
    const int                   nthreads,
    const int                   G,
    const int                   D,
    const int                   S,
    const Tx*                   x,
    const Tp*                   mu,
    const Tp*                   rsig,
    const Tp*                   gamma,
    const Tp*                   ds,
    const Tp*                   db,
    const Tx*                   dy,
    Tx*                         dx) {
    const int C = G * D;
    const Tp denom = Tp(1) / static_cast<Tp>(D * S);
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        const int i_mu = kOrder == StorageOrder::NCHW ?
            i / (D * S) : i / (C * S) * G + (i / D % G);
        const int i_gamma = kOrder == StorageOrder::NCHW ?
            (i / S) % C : i % C;
#if __CUDA_ARCH__ >= 350
        const Tp u = (
            L(db, i_mu) * L(mu, i_mu) - L(ds, i_mu))
                * (L(x, i) - L(mu, i_mu))
                    * utils::math::Cube<Tp>(L(rsig, i_mu));
        const Tp v = L(db, i_mu) * L(rsig, i_mu);
        dx[i] = L(gamma, i_gamma) * L(dy, i)
            * L(rsig, i_mu) + (u - v) * denom;
#else
        const Tp u = (
            db[i_mu] * mu[i_mu] - ds[i_mu])
                * (x[i] - mu[i_mu])
                    * utils::math::Cube<Tp>(rsig[i_mu]);
        const Tp v = db[i_mu] * rsig[i_mu];
        dx[i] = gamma[i_gamma] * dy[i]
            * rsig[i_mu] + (u - v) * denom;
#endif
    }
}

template <StorageOrder kOrder>
__global__ void _GroupNormGradHalf(
    const int                   nthreads,
    const int                   G,
    const int                   D,
    const int                   S,
    const half*                 x,
    const float*                mu,
    const float*                rsig,
    const float*                gamma,
    const float*                ds,
    const float*                db,
    const half*                 dy,
    half*                       dx) {
#if __CUDA_ARCH__ >= 530
    const int C = G * D;
    const float denom = 1.f / static_cast<float>(D * S);
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        const int i_mu = kOrder == StorageOrder::NCHW ?
            i / (D * S) : i / (C * S) * G + (i / D % G);
        const int i_gamma = kOrder == StorageOrder::NCHW ?
            (i / S) % C : i % C;
        const float u = (
            L(db, i_mu) * L(mu, i_mu) - L(ds, i_mu))
                * (LF(x, i) - L(mu, i_mu))
                    * utils::math::Cube<float>(L(rsig, i_mu));
        const float v = L(db, i_mu) * L(rsig, i_mu);
        dx[i] = __float2half(
            L(gamma, i_gamma) * LF(dy, i)
                * L(rsig, i_mu) + (u - v) * denom
        );
    }
#endif
}

/*! Kernel Launchers */

#define DEFINE_FORWARD_KERNEL_LAUNCHER(Tx, Tp) \
    template <> void GroupNormForward<Tx, Tp, CUDAContext>( \
        const int                   N, \
        const int                   G, \
        const int                   D, \
        const int                   S, \
        const string&               data_format, \
        const Tx*                   x, \
        const Tp*                   mu, \
        const Tp*                   rsig, \
        const Tp*                   gamma, \
        const Tp*                   beta, \
        Tp*                         scale, \
        Tp*                         bias, \
        Tx*                         y, \
        CUDAContext*                ctx) { \
        const int C = G * D; \
        _GroupNormFusedParams<Tp> \
            << < CUDA_2D_BLOCKS(N * G), CUDA_THREADS, \
                 0, ctx->cuda_stream() >> > \
            (N, G, D, mu, rsig, gamma, beta, scale, bias); \
        if (data_format == "NCHW") { \
            _GroupNormForwardNCHW<Tx, Tp> \
                << < CUDA_2D_BLOCKS(N * C), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (N, C, S, x, scale, bias, y); \
        } else if (data_format == "NHWC") { \
            _GroupNormForwardNHWC<Tx, Tp> \
                << < CUDA_2D_BLOCKS(N * C), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (N, C, S, x, scale, bias, y); \
        } \
    }

#define DEFINE_BACKWARD_KERNEL_LAUNCHER(Tx, Tp) \
    template <> void GroupNormBackward<Tx, Tp, CUDAContext>( \
        const int                   N, \
        const int                   G, \
        const int                   D, \
        const int                   S, \
        const string&               data_format, \
        const Tx*                   x, \
        const Tp*                   mu, \
        const Tp*                   rsig, \
        const Tp*                   gamma, \
        const Tx*                   dy, \
        Tp*                         ds, \
        Tp*                         db, \
        Tx*                         dx, \
        Tp*                         dgamma, \
        Tp*                         dbeta, \
        CUDAContext*                ctx) { \
        auto nthreads = N * G * D * S; \
        if (data_format == "NCHW") { \
            _GroupNormWGrad<Tx, Tp, StorageOrder::NCHW> \
                << < CUDA_2D_BLOCKS(G * D), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (N, G, D, S, x, mu, rsig, dy, dgamma, dbeta); \
            _GroupNormInternalGrad<Tx, Tp, StorageOrder::NCHW> \
                << < CUDA_2D_BLOCKS(N * G), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (N, G, D, S, x, gamma, dy, ds, db); \
            _GroupNormGrad<Tx, Tp, StorageOrder::NCHW> \
                << < CUDA_BLOCKS(nthreads), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (nthreads, G, D, S, x, mu, rsig, gamma, ds, db, dy, dx); \
        } else if (data_format == "NHWC") { \
            _GroupNormWGrad<Tx, Tp, StorageOrder::NHWC> \
                << < CUDA_2D_BLOCKS(G * D), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (N, G, D, S, x, mu, rsig, dy, dgamma, dbeta); \
            _GroupNormInternalGrad<Tx, Tp, StorageOrder::NHWC> \
                << < CUDA_2D_BLOCKS(N * G), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (N, G, D, S, x, gamma, dy, ds, db); \
            _GroupNormGrad<Tx, Tp, StorageOrder::NHWC> \
                << < CUDA_BLOCKS(nthreads), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (nthreads, G, D, S, x, mu, rsig, gamma, ds, db, dy, dx); \
        } \
    }

DEFINE_FORWARD_KERNEL_LAUNCHER(float, float);
DEFINE_BACKWARD_KERNEL_LAUNCHER(float, float);

template <> void GroupNormForward<float16, float, CUDAContext>(
    const int                   N,
    const int                   G,
    const int                   D,
    const int                   S,
    const string&               data_format,
    const float16*              x,
    const float*                mu,
    const float*                rsig,
    const float*                gamma,
    const float*                beta,
    float*                      scale,
    float*                      bias,
    float16*                    y,
    CUDAContext*                ctx) {
    const int C = G * D;
    _GroupNormFusedParams<float>
        << < CUDA_2D_BLOCKS(N * G), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (N, G, D, mu, rsig, gamma, beta, scale, bias);
    if (data_format == "NCHW") {
        _GroupNormForwardNCHW<half, float>
            << < CUDA_2D_BLOCKS(N * C), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (N, C, S, reinterpret_cast<const half*>(x),
                scale, bias, reinterpret_cast<half*>(y));
    } else if (data_format == "NHWC") {
        _GroupNormForwardNHWC<half, float>
            << < CUDA_2D_BLOCKS(N * C), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (N, C, S, reinterpret_cast<const half*>(x),
                scale, bias, reinterpret_cast<half*>(y));
    }
}

template <> void GroupNormBackward<float16, float, CUDAContext>(
    const int                   N,
    const int                   G,
    const int                   D,
    const int                   S,
    const string&               data_format,
    const float16*              x,
    const float*                mu,
    const float*                rsig,
    const float*                gamma,
    const float16*              dy,
    float*                      ds,
    float*                      db,
    float16*                    dx,
    float*                      dgamma,
    float*                      dbeta,
    CUDAContext*                ctx) {
    auto nthreads = N * G * D * S;
    if (data_format == "NCHW") {
        _GroupNormWGradHalf<StorageOrder::NCHW>
            << < CUDA_2D_BLOCKS(G * D), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (N, G, D, S, reinterpret_cast<const half*>(x),
                mu, rsig, reinterpret_cast<const half*>(dy),
                    dgamma, dbeta);
        _GroupNormInternalGradHalf<StorageOrder::NCHW>
            << < CUDA_2D_BLOCKS(N * G), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (N, G, D, S, reinterpret_cast<const half*>(x),
                gamma, reinterpret_cast<const half*>(dy),
                    ds, db);
        _GroupNormGradHalf<StorageOrder::NCHW>
            << < CUDA_BLOCKS(nthreads), CUDA_THREADS,
                 0, ctx->cuda_stream() >> > \
            (nthreads, G, D, S, reinterpret_cast<const half*>(x),
                mu, rsig, gamma, ds, db,
                    reinterpret_cast<const half*>(dy),
                        reinterpret_cast<half*>(dx));
    } else if (data_format == "NHWC") { \
        _GroupNormWGradHalf<StorageOrder::NHWC>
            << < CUDA_2D_BLOCKS(G * D), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (N, G, D, S, reinterpret_cast<const half*>(x),
                mu, rsig, reinterpret_cast<const half*>(dy),
                    dgamma, dbeta);
        _GroupNormInternalGradHalf<StorageOrder::NHWC>
            << < CUDA_2D_BLOCKS(N * G), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (N, G, D, S, reinterpret_cast<const half*>(x),
                gamma, reinterpret_cast<const half*>(dy),
                    ds, db);
        _GroupNormGradHalf<StorageOrder::NHWC>
            << < CUDA_BLOCKS(nthreads), CUDA_THREADS,
                 0, ctx->cuda_stream() >> > \
            (nthreads, G, D, S, reinterpret_cast<const half*>(x),
                mu, rsig, gamma, ds, db,
                    reinterpret_cast<const half*>(dy),
                        reinterpret_cast<half*>(dx));
    }
}

#undef L
#undef LF
#undef DEFINE_FORWARD_KERNEL_LAUNCHER
#undef DEFINE_BACKWARD_KERNEL_LAUNCHER

}  // namespace kernel

}  // namespace dragon

#endif  // WITH_CUDA