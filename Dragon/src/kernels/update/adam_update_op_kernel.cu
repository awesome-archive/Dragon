
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/cast.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/*! AdamUpdate <T = float32, Device = CUDA> */

template <typename T>
__global__ void _AdamUpdate(
    const int               count,
    const T                 lr,
    const T                 beta1,
    const T                 beta2,
    const T                 eps,
    T*                      g,
    T*                      m,
    T*                      v) {
    CUDA_1D_KERNEL_LOOP(i, count) {
        T gi = g[i];
        T mi = m[i] = m[i] * beta1 + gi * (1 - beta1);
        T vi = v[i] = v[i] * beta2 + gi * gi * (1 - beta2);
        g[i] = lr * mi / (sqrt(vi) + eps);
    }
}

template <> void AdamUpdate<float, CUDAContext>(
    const int               count,
    const float             lr,
    const float             beta1,
    const float             beta2,
    const float             eps,
    float*                  g,
    float*                  m,
    float*                  v,
    CUDAContext*            ctx) {
    _AdamUpdate<float>
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (count, lr, beta1, beta2, eps, g, m, v);
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA