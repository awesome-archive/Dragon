
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/cast.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/* <T = float16, Device = CUDA> */

__global__ void _MixedPrecL2DecayHalf(
    const int               nthreads,
    const float             alpha,
    const half*             w,
    float*                  dx) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
        dx[i] += (__half2float(w[i]) * alpha);
#endif
    }
}

template <> void MixedPrecL2Decay<float16, CUDAContext>(
    const int               count,
    const float             alpha,
    const float16*          w,
    float*                  dx,
    CUDAContext*            ctx) {
    _MixedPrecL2DecayHalf
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        count,
        alpha,
        reinterpret_cast<const half*>(w),
        dx
    );
}

/* <T = float16, Device = CUDA> */

__global__ void _MixedPrecUpdateHalf(
    const int               nthreads,
    const float*            updates,
    half*                   w) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
        w[i] = __float2half(__half2float(
            w[i]) - updates[i]);
#endif
    }
}

template <> void MixedPrecUpdate<float16, CUDAContext>(
    const int               count,
    const float*            updates,
    float16*                w,
    CUDAContext*            ctx) {
    _MixedPrecUpdateHalf
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        count,
        updates,
        reinterpret_cast<half*>(w)
    );
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA