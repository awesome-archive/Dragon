
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/cast.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/* <T = float32, Device = CUDA> */

template <typename T>
__global__ void _RMSPropUpdate(
    const int               nthreads,
    const T                 lr,
    const T                 decay,
    const T                 eps,
    T*                      g,
    T*                      h) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
        T gi = g[i];
        T hi = h[i] = decay * h[i] + (1 - decay) * gi * gi;
        g[i] = lr * g[i] / (sqrt(hi) + eps);
    }
}

template <> void RMSPropUpdate<float, CUDAContext>(
    const int               count,
    const float             lr,
    const float             decay,
    const float             eps,
    float*                  g,
    float*                  h,
    CUDAContext*            ctx) {
    _RMSPropUpdate
        << < CUDA_BLOCKS(count), CUDA_THREADS,
             0, ctx->cuda_stream() >> >(
        count, lr, decay, eps, g, h
    );
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA