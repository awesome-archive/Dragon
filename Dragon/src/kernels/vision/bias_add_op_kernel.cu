
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/op_kernel.h"

namespace dragon {

namespace kernel {

/*! BiasAdd <T = float32, Device = CUDA> */

template <typename T>
__global__ void _BiasAdd_NCHW(
    const int               nthreads,
    const int               dim,
    const int               inner_dim,
    const T*                bias,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 350
        y[i] += __ldg(bias + ((i / inner_dim) % dim));
#else
        y[i] += bias[(i / inner_dim) % dim];
#endif
    }
}

template <typename T>
__global__ void _BiasAdd_NHWC(
    const int               nthreads,
    const int               dim,
    const int               inner_dim,
    const T*                bias,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 350
        y[i] += __ldg(bias + (i % dim));
#else
        y[i] += bias[i % dim];
#endif
    }
}

template<> void BiasAdd<float, CUDAContext>(
    const int               outer_dim,
    const int               dim,
    const int               inner_dim,
    const string&           data_format,
    const float*            bias,
    const float*            bias_multiplier,
    float*                  y,
    CUDAContext*            ctx) {
    auto nthreads = outer_dim * dim * inner_dim;
    if (data_format == "NCHW") {
        _BiasAdd_NCHW<float>
            << < CUDA_BLOCKS(nthreads), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (nthreads, dim, inner_dim, bias, y);
    } else if (data_format == "NHWC") {
        _BiasAdd_NHWC<float>
            << < CUDA_BLOCKS(nthreads), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (nthreads, dim, inner_dim, bias, y);
    } else LOG(FATAL) << "Unknown data format: " << data_format;
}

}  // namespace kernel

}  // namepsace dragon

#endif  // WITH_CUDA