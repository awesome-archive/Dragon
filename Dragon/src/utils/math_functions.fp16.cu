
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include "core/context_cuda.h"
#include "utils/cast.h"
#include "utils/op_kernel.h"
#include "utils/math_functions.h"

namespace dragon {

namespace math {

/*!
 * ----------------------------------------------
 *
 *
 *            Simple Unary Functions
 *
 *
 * ----------------------------------------------
 */

template <typename T>
__global__ void _ExpHalf(
    const int               n,
    const T*                a,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = hexp(a[i]);
#endif
    }
}

template <typename T>
__global__ void _ExpHalf2(
    const int               n,
    const T*                a,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = h2exp(a[i]);
#endif
    }
}

template <> void Exp<float16, CUDAContext>(
    int                     n,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    if ((n & 1) == 0) {
        _ExpHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n >> 1,
                    reinterpret_cast<const half2*>(x),
                        reinterpret_cast<half2*>(y));
    }
    else {
        _ExpHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n,
                     reinterpret_cast<const half*>(x),
                        reinterpret_cast<half*>(y));
    }
}

template <typename T>
__global__ void _LogHalf(
    const int               n,
    const T*                a,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = hlog(a[i]);
#endif
    }
}

template <typename T>
__global__ void _LogHalf2(
    const int               n,
    const T*                a,
    T*                      y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = h2log(a[i]);
#endif
    }
}

template <> void Log<float16, CUDAContext>(
    int                     n,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    if ((n & 1) == 0) {
        _LogHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n >> 1,
                    reinterpret_cast<const half2*>(x),
                        reinterpret_cast<half2*>(y));
    }
    else {
        _LogHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n,
                     reinterpret_cast<const half*>(x),
                        reinterpret_cast<half*>(y));
    }
}

template <typename T>
__global__ void _InvHalf(
    const int               n,
    const half*             x,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] =  hrcp(x[i]);
#endif
    }
}

template <typename T>
__global__ void _InvHalf2(
    const int               n,
    const half2*            x,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = h2rcp(x[i]);
#endif
    }
}

template <> void Inv<float16, CUDAContext>(
    const int               n,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    if ((n & 1) == 0) {
        _InvHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n >> 1,
                     reinterpret_cast<const half2*>(x),
                        reinterpret_cast<half2*>(y));
    } else {
        _InvHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n,
                     reinterpret_cast<const half*>(x),
                        reinterpret_cast<half*>(y));
    }
}

template <typename T>
__global__ void _SqrtHalf(
    int                     n,
    const half*             x,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = hsqrt(x[i]);
#endif
    }
}

template <typename T>
__global__ void _SqrtHalf2(
    const int               n,
    const half2*            x,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = h2sqrt(x[i]);
#endif
    }
}

template <> void Sqrt<float16, CUDAContext>(
    int                     n,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    if ((n & 1) == 0) {
        _SqrtHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n >> 1,
                     reinterpret_cast<const half2*>(x),
                         reinterpret_cast<half2*>(y));
    } else {
        _SqrtHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n,
                     reinterpret_cast<const half*>(x),
                         reinterpret_cast<half*>(y));
    }
}

template <typename T>
__global__ void _RSqrtHalf(
    int                     n,
    const half*             x,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = hrsqrt(x[i]);
#endif
    }
}

template <typename T>
__global__ void _RSqrtHalf2(
    const int               n,
    const half2*            x,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = h2rsqrt(x[i]);
#endif
    }
}

template <> void RSqrt<float16, CUDAContext>(
    int                     n,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    if ((n & 1) == 0) {
        _RSqrtHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n >> 1,
                     reinterpret_cast<const half2*>(x),
                         reinterpret_cast<half2*>(y));
    } else {
        _RSqrtHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n,
                     reinterpret_cast<const half*>(x),
                         reinterpret_cast<half*>(y));
    }
}

template <typename T>
__global__ void _SquareHalf(
    const int               n,
    const half*             x,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hmul(x[i], x[i]);
#endif
    }
}

template <typename T>
__global__ void _SquareHalf2(
    const int               n,
    const half2*            x,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hmul2(x[i], x[i]);
#endif
    }
}

template <> void Square<float16, CUDAContext>(
    int                     n,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    if ((n & 1) == 0) {
        _SquareHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n >> 1,
                     reinterpret_cast<const half2*>(x),
                         reinterpret_cast<half2*>(y));
    } else {
        _SquareHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n,
                     reinterpret_cast<const half*>(x),
                         reinterpret_cast<half*>(y));
    }
}

/*!
 * ----------------------------------------------
 *
 *
 *             Scale Unary Functions
 *
 *
 * ----------------------------------------------
 */

/*!                y = a                 */

template <typename T>
__global__ void _SetHalf(
    const int               n,
    const T                 alpha,
    T*                      x) {
    CUDA_1D_KERNEL_LOOP(i, n) {
        x[i] = alpha;
    }
}

template <> void Set<float16, CUDAContext>(
    const int               n,
    const float16           alpha,
    float16*                y,
    CUDAContext*            ctx) {
    if (alpha.x == (unsigned short)0) {
        CUDA_CHECK(cudaMemsetAsync(y, 0,
            sizeof(float16) * n, ctx->cuda_stream()));
        return;
    }
    if ((n & 1) == 0) {
        _SetHalf<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n >> 1,
                     cast::to<half2>(alpha),
                         reinterpret_cast<half2*>(y));
    } else {
        _SetHalf<float16>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n, alpha, y);
    }
}

/*!                y = x^e                */

template <typename T>
__global__ void _PowHalf(
    const int               n,
    const float             alpha,
    const half*             a,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hmul(a[i], a[i]);
#endif
    }
}

template <typename T>
__global__ void _PowHalf2(
    const int               n,
    const float             alpha,
    const half2*            a,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hmul2(a[i], a[i]);
#endif
    }
}

template <> void Pow<float16, CUDAContext>(
    int                     n,
    const float             alpha,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    CHECK(alpha == float(2)) << "\nRequired power = 2";
    if ((n & 1) == 0) {
        _PowHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n >> 1,
                     alpha, reinterpret_cast<const half2*>(x),
                         reinterpret_cast<half2*>(y));
    } else {
        _PowHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >(n,
                     alpha, reinterpret_cast<const half*>(x),
                         reinterpret_cast<half*>(y));
    }
}

/*!        y = ax    ||    x = ax        */

template <> void Scale<float16, CUDAContext>(
    const int               n,
    const float             alpha,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    if (x != y) {
        CUDA_CHECK(cudaMemcpyAsync(y, x, sizeof(float16) * n,
            cudaMemcpyDeviceToDevice, ctx->cuda_stream()));
    }
    if (alpha != 1.f) {
        CUBLAS_CHECK(cublasScalEx(
            ctx->cublas_handle(), n,
                &alpha, CUDA_R_32F,
                    y, CUDA_R_16F, 1,
                        CUDA_R_32F));
    }
}

/*!                y += ax                */

template <> void Axpy<float16, CUDAContext>(
    const int               n,
    const float             alpha,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    CUBLAS_CHECK(cublasAxpyEx(
        ctx->cublas_handle(), n,
            &alpha, CUDA_R_32F,
                x, CUDA_R_16F, 1,
                    y, CUDA_R_16F, 1,
                        CUDA_R_32F));
}

/*!                 y = ax + by               */

template <> void Axpby<float16, CUDAContext>(
    const int               n,
    const float             alpha,
    const float16*          x,
    const float             beta,
    float16*                y,
    CUDAContext*            ctx) {
    Scale(n, beta, y, y, ctx);
    Axpy(n, alpha, x, y, ctx);
}

/*!                 y += a                */

template <typename T>
__global__ void _AddScalarHalf(
    const int               n,
    half                    alpha,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hadd(y[i], alpha);
#endif
    }
}

template <typename T>
__global__ void _AddScalarHalf2(
    const int               n,
    half2                   alpha,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hadd2(y[i], alpha);
#endif
    }
}

template <> void AddScalar<float16, CUDAContext>(
    const int               n,
    const float             alpha,
    float16*                y,
    CUDAContext*            ctx) {
    if ((n & 1) == 0) {
        _AddScalarHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (n >> 1, cast::to<half2>(alpha),
                reinterpret_cast<half2*>(y));
    } else {
        _AddScalarHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (n, cast::to<half>(alpha),
                reinterpret_cast<half*>(y));
    }
}

/*!
 * ----------------------------------------------
 *
 *
 *             Extended Unary Functions
 *
 *
 * ----------------------------------------------
 */

template <typename T>
__global__ void _InvStdHalf(
    int                     n,
    const half              eps,
    const half*             x,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = hrsqrt(__hadd(x[i], eps));
#endif
    }
}

template <typename T>
__global__ void _InvStdHalf2(
    const int               n,
    const half2             eps,
    const half2*            x,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = h2rsqrt(__hadd2(x[i], eps));
#endif
    }
}

template <> void InvStd<float16, CUDAContext>(
    int                     n,
    const float             eps,
    const float16*          x,
    float16*                y,
    CUDAContext*            ctx) {
    if ((n & 1) == 0) {
        _InvStdHalf2<half2>
            << < CUDA_BLOCKS(n >> 1), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (n >> 1, cast::to<half2>(eps),
                reinterpret_cast<const half2*>(x),
                    reinterpret_cast<half2*>(y));
    } else {
        _InvStdHalf<half>
            << < CUDA_BLOCKS(n), CUDA_THREADS,
                 0, ctx->cuda_stream() >> >
            (n, cast::to<half>(eps),
                reinterpret_cast<const half*>(x),
                    reinterpret_cast<half*>(y));
    }
}

/*!
 * ----------------------------------------------
 *
 *
 *            Simply Binary Functions
 *
 *
 * ----------------------------------------------
 */

__global__ void _AddHalf(
    const int               n,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hadd(a[i], b[i]);
#endif
    }
}

__global__ void _AddHalf2(
    const int               n,
    const half2*            a,
    const half2*            b,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hadd2(a[i], b[i]);
#endif
    }
}

__global__ void _SubHalf(
    const int               n,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hsub(a[i], b[i]);
#endif
    }
}

__global__ void _SubHalf2(
    const int               n,
    const half2*            a,
    const half2*            b,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hsub2(a[i], b[i]);
#endif
    }
}

__global__ void _MulHalf(
    const int               n,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hmul(a[i], b[i]);
#endif
    }
}

__global__ void _MulHalf2(
    const int               n,
    const half2*            a,
    const half2*            b,
    half2*                  y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hmul2(a[i], b[i]);
#endif
    }
}

__global__ void _DivHalf(
    const int               n,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(i, n) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hdiv(a[i], b[i]);
#endif
    }
}

#define DEFINE_SIMPLE_BINARY_FUNC(name) \
    template <> void name<float16, CUDAContext>( \
        const int               n, \
        const float16*          a, \
        const float16*          b, \
        float16*                y, \
        CUDAContext*            ctx) { \
        if ((n & 1) == 0) { \
            _##name##Half2 \
                << < CUDA_BLOCKS(n >> 1), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (n >> 1, reinterpret_cast<const half2*>(a), \
                    reinterpret_cast<const half2*>(b), \
                        reinterpret_cast<half2*>(y)); \
        } else { \
            _##name##Half \
                << < CUDA_BLOCKS(n), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (n, reinterpret_cast<const half*>(a), \
                    reinterpret_cast<const half*>(b), \
                        reinterpret_cast<half*>(y)); \
        } \
    }

DEFINE_SIMPLE_BINARY_FUNC(Add);
DEFINE_SIMPLE_BINARY_FUNC(Sub);
DEFINE_SIMPLE_BINARY_FUNC(Mul);
#undef DEFINE_SIMPLE_BINARY_FUNC

template <> void Div<float16, CUDAContext>(
    int                     n,
    const float16*          a,
    const float16*          b,
    float16*                y,
    CUDAContext*            ctx) {
    _DivHalf
        << < CUDA_BLOCKS(n), CUDA_THREADS,
             0, ctx->cuda_stream() >> >
        (n, reinterpret_cast<const half*>(a),
            reinterpret_cast<const half*>(b),
                reinterpret_cast<half*>(y));
}

template <> void Dot<float16, CUDAContext>(
    int                     n,
    const float16*          a,
    const float16*          b,
    float16*                y,
    CUDAContext*            ctx) {
    CUBLAS_CHECK(cublasDotEx(
        ctx->cublas_handle(), n,
            a, CUDA_R_16F, 1,
                b, CUDA_R_16F, 1,
                    y, CUDA_R_16F,
                        CUDA_R_32F));
    ctx->FinishDeviceCompution();
}

/*!
 * ----------------------------------------------
 *
 *
 *          Broadcast Binary Functions
 *
 *
 * ----------------------------------------------
 */

template <bool BroadcastA>
__global__ void _RowBroadcastAddHalf(
    const int               count,
    const int               cols,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int i = idx % cols;
        const int a_idx = BroadcastA ? i : idx;
        const int b_idx = BroadcastA ? idx : i;
        y[idx] = __hadd(a[a_idx], b[b_idx]);
#endif
    }
}

template <bool BroadcastA>
__global__ void _ColBroadcastAddHalf(
    const int               count,
    const int               cols,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int i = idx / cols;
        const int a_idx = BroadcastA ? i : idx;
        const int b_idx = BroadcastA ? idx : i;
        y[idx] = __hadd(a[a_idx], b[b_idx]);
#endif
    }
}

template <bool BroadcastA>
__global__ void _RowBroadcastSubHalf(
    const int               count,
    const int               cols,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int i = idx % cols;
        const int a_idx = BroadcastA ? i : idx;
        const int b_idx = BroadcastA ? idx : i;
        y[idx] = __hsub(a[a_idx], b[b_idx]);
#endif
    }
}

template <bool BroadcastA>
__global__ void _ColBroadcastSubHalf(
    const int               count,
    const int               cols,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int i = idx / cols;
        const int a_idx = BroadcastA ? i : idx;
        const int b_idx = BroadcastA ? idx : i;
        y[idx] = __hsub(a[a_idx], b[b_idx]);
#endif
    }
}

template <bool BroadcastA>
__global__ void _RowBroadcastMulHalf(
    const int               count,
    const int               cols,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int i = idx % cols;
        const int a_idx = BroadcastA ? i : idx;
        const int b_idx = BroadcastA ? idx : i;
        y[idx] = __hmul(a[a_idx], b[b_idx]);
#endif
    }
}

template <bool BroadcastA>
__global__ void _ColBroadcastMulHalf(
    const int               count,
    const int               cols,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int i = idx / cols;
        const int a_idx = BroadcastA ? i : idx;
        const int b_idx = BroadcastA ? idx : i;
        y[idx] = __hmul(a[a_idx], b[b_idx]);
#endif
    }
}

template <bool BroadcastA>
__global__ void _RowBroadcastDivHalf(
    const int               count,
    const int               cols,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int i = idx % cols;
        const int a_idx = BroadcastA ? i : idx;
        const int b_idx = BroadcastA ? idx : i;
        y[idx] = __hdiv(a[a_idx], b[b_idx]);
#endif
    }
}

template <bool BroadcastA>
__global__ void _ColBroadcastDivHalf(
    const int               count,
    const int               cols,
    const half*             a,
    const half*             b,
    half*                   y) {
    CUDA_1D_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        const int i = idx / cols;
        const int a_idx = BroadcastA ? i : idx;
        const int b_idx = BroadcastA ? idx : i;
        y[idx] = __hdiv(a[a_idx], b[b_idx]);
#endif
    }
}

#define DEFINE_BROADCAST_BINARY_FUNC(name) \
    template <> void Broadcast##name<float16, CUDAContext>( \
        const int               rows, \
        const int               cols, \
        const int               type, \
        const float16*          a, \
        const float16*          b, \
        float16*                y, \
        CUDAContext*            ctx) { \
        auto n = rows * cols; \
        if (type == 0) { \
            /*! Row - BroadcastB */ \
            _RowBroadcast##name##Half<false> \
                << < CUDA_BLOCKS(n), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (n, cols, reinterpret_cast<const half*>(a), \
                    reinterpret_cast<const half*>(b), \
                        reinterpret_cast<half*>(y)); \
        } else if (type == 1) { \
            /*! Col - BroadcastB */ \
            _ColBroadcast##name##Half<false> \
                << < CUDA_BLOCKS(n), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (n, cols, reinterpret_cast<const half*>(a), \
                    reinterpret_cast<const half*>(b), \
                        reinterpret_cast<half*>(y)); \
        } else if (type == 2) { \
            /*! Row - BroadcastA */ \
            _RowBroadcast##name##Half<true> \
                << < CUDA_BLOCKS(n), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (n, cols, reinterpret_cast<const half*>(a), \
                    reinterpret_cast<const half*>(b), \
                        reinterpret_cast<half*>(y)); \
        } else if (type == 3) { \
            /*! Col - BroadcastA */ \
            _ColBroadcast##name##Half<true> \
                << < CUDA_BLOCKS(n), CUDA_THREADS, \
                     0, ctx->cuda_stream() >> > \
                (n, cols, reinterpret_cast<const half*>(a), \
                    reinterpret_cast<const half*>(b), \
                        reinterpret_cast<half*>(y)); \
        } else { \
            LOG(FATAL) << "Unknown broadcast type: " << type; \
        } \
    }

DEFINE_BROADCAST_BINARY_FUNC(Add);
DEFINE_BROADCAST_BINARY_FUNC(Sub);
DEFINE_BROADCAST_BINARY_FUNC(Mul);
DEFINE_BROADCAST_BINARY_FUNC(Div);
#undef DEFINE_BROADCAST_BINARY_FUNC

/*!
 * ----------------------------------------------
 *
 *
 *        Linear Algebra Binary Functions
 *
 *
 * ----------------------------------------------
 */

template <> void Gemm<float16, CUDAContext>(
    const CBLAS_TRANSPOSE   TransA,
    const CBLAS_TRANSPOSE   TransB,
    const int               M,
    const int               N,
    const int               K,
    const float             alpha,
    const float16*          A,
    const float16*          B,
    const float             beta,
    float16*                C,
    CUDAContext*            ctx,
    TensorProto_DataType    math_type) {
    int lda = (TransA == CblasNoTrans) ? K : M;
    int ldb = (TransB == CblasNoTrans) ? N : K;
    cublasOperation_t cuTransA = (TransA == CblasNoTrans) ?
        CUBLAS_OP_N : CUBLAS_OP_T;
    cublasOperation_t cuTransB = (TransB == CblasNoTrans) ?
        CUBLAS_OP_N : CUBLAS_OP_T;
    if (math_type == TensorProto_DataType_FLOAT) {
        const float _alpha_ = alpha, _beta_ = beta;
#if CUDA_VERSION >= 9000
        if (TENSOR_CORE_AVAILABLE()) {
            //  GEMM + MATH32 + TENSOR-CORE
            CUBLAS_CHECK(cublasGemmEx(
                ctx->cublas_handle(),
                cuTransB, cuTransA, N, M, K,
                &_alpha_,
                    B, CUDA_R_16F, ldb,
                    A, CUDA_R_16F, lda,
                &_beta_,
                    C, CUDA_R_16F, N,
                CUDA_R_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        } else {
            //  GEMM + MATH32 + DEFAULT
            CUBLAS_CHECK(cublasSgemmEx(
                ctx->cublas_handle(),
                cuTransB, cuTransA, N, M, K,
                &_alpha_,
                    B, CUDA_R_16F, ldb,
                    A, CUDA_R_16F, lda,
                &_beta_,
                    C, CUDA_R_16F, N));
        }
#else
       CUBLAS_CHECK(cublasSgemmEx(
           ctx->cublas_handle(),
           cuTransB, cuTransA, N, M, K,
           &_alpha_,
               B, CUDA_R_16F, ldb,
               A, CUDA_R_16F, lda,
           &_beta_,
               C, CUDA_R_16F, N));
#endif
    } else if (math_type == TensorProto_DataType_FLOAT16) {
        const half _alpha_ = cast::to<half>(alpha);
        const half _beta_ = cast::to<half>(beta);
#if CUDA_VERSION >= 9000
        if (TENSOR_CORE_AVAILABLE()) {
            //  GEMM + MATH16 + TENSOR-CORE
            CUBLAS_CHECK(cublasGemmEx(
                ctx->cublas_handle(),
                cuTransB, cuTransA, N, M, K,
                &_alpha_,
                    B, CUDA_R_16F, ldb,
                    A, CUDA_R_16F, lda,
                &_beta_,
                    C, CUDA_R_16F, N,
                CUDA_R_16F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        } else {
            //  GEMM + MATH16 + DEFAULT
            CUBLAS_CHECK(cublasHgemm(
                ctx->cublas_handle(),
                cuTransB, cuTransA, N, M, K,
                &_alpha_,
                    reinterpret_cast<const half*>(B), ldb,
                    reinterpret_cast<const half*>(A), lda,
                &_beta_,
                    reinterpret_cast<half*>(C), N));
        }
#else
        CUBLAS_CHECK(cublasHgemm(
            ctx->cublas_handle(),
            cuTransB, cuTransA, N, M, K,
            &_alpha_,
                reinterpret_cast<const half*>(B), ldb,
                reinterpret_cast<const half*>(A), lda,
            &_beta_,
                reinterpret_cast<half*>(C), N));
#endif
    } else {
        LOG(FATAL) << "Unsupported math type";
    }
}

template <> void Gemv<float16, CUDAContext>(
    const CBLAS_TRANSPOSE   TransA,
    const int               M,
    const int               N,
    const float             alpha,
    const float16*          A,
    const float16*          x,
    const float             beta,
    float16*                y,
    CUDAContext*            ctx,
    TensorProto_DataType    math_type) {
    cublasOperation_t cuTransA = (TransA == CblasNoTrans) ?
        CUBLAS_OP_T : CUBLAS_OP_N;
    int m = (cuTransA == CUBLAS_OP_N) ? N : M;
    int k = (cuTransA == CUBLAS_OP_N) ? M : N;
    int LDA = (cuTransA == CUBLAS_OP_N) ? m : k;
    int LDC = m;
    const float _alpha_ = alpha, _beta_ = beta;
    if (math_type == TensorProto_DataType_FLOAT) {
#if CUDA_VERSION >= 9000
        if (TENSOR_CORE_AVAILABLE()) {
            //  GEMV + MATH32 + TENSOR-CORE
            CUBLAS_CHECK(cublasGemmEx(
                ctx->cublas_handle(),
                cuTransA, CUBLAS_OP_N, m, 1, k,
                &_alpha_,
                    A, CUDA_R_16F, LDA,
                    x, CUDA_R_16F, k,
                &_beta_,
                    y, CUDA_R_16F, LDC,
                CUDA_R_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        } else {
            //  GEMV + MATH32 + DEFAULT
            CUBLAS_CHECK(cublasSgemmEx(
                ctx->cublas_handle(),
                cuTransA, CUBLAS_OP_N, m, 1, k,
                &_alpha_,
                    A, CUDA_R_16F, LDA,
                    x, CUDA_R_16F, k,
                &_beta_,
                    y, CUDA_R_16F, LDC));
        }
#else
        CUBLAS_CHECK(cublasSgemmEx(
            ctx->cublas_handle(),
            cuTransA, CUBLAS_OP_N, m, 1, k,
            &_alpha_,
                A, CUDA_R_16F, LDA,
                x, CUDA_R_16F, k,
            &_beta_,
                y, CUDA_R_16F, LDC));
#endif
    } else if (math_type == TensorProto_DataType_FLOAT16) {
        const half _alpha_ = cast::to<half>(alpha);
        const half _beta_ = cast::to<half>(beta);
#if CUDA_VERSION >= 9000
        if (TENSOR_CORE_AVAILABLE()) {
            //  GEMV + MATH16 + TENSOR-CORE
            CUBLAS_CHECK(cublasGemmEx(
                ctx->cublas_handle(),
                cuTransA, CUBLAS_OP_N, m, 1, k,
                &_alpha_,
                    A, CUDA_R_16F, LDA,
                    x, CUDA_R_16F, k,
                &_beta_,
                    y, CUDA_R_16F, LDC,
                CUDA_R_16F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        } else {
            //  GEMV + MATH16 + DEFAULT
            CUBLAS_CHECK(cublasHgemm(
                ctx->cublas_handle(),
                cuTransA, CUBLAS_OP_N, m, 1, k,
                &_alpha_,
                    reinterpret_cast<const half*>(A), LDA,
                    reinterpret_cast<const half*>(x), k,
                &_beta_,
                    reinterpret_cast<half*>(y), LDC));
        }
#else
        CUBLAS_CHECK(cublasHgemm(
            ctx->cublas_handle(),
            cuTransA, CUBLAS_OP_N, m, 1, k,
            &_alpha_,
                reinterpret_cast<const half*>(A), LDA,
                reinterpret_cast<const half*>(x), k,
            &_beta_,
                reinterpret_cast<half*>(y), LDC));
#endif
    } else {
            LOG(FATAL) << "Unsupported math type";
    }
}

/*!
 * ----------------------------------------------
 *
 *
 *               Random Functions
 *
 *
 * ----------------------------------------------
 */

template <> void RandomUniform<float16, CUDAContext>(
    const int               n,
    const float             low,
    const float             high,
    float16*                y,
    CUDAContext*            ctx) {
    auto* y32 = (float*)ctx->New(n * sizeof(float));
    math::RandomUniform<float, CUDAContext>(n, low, high, y32, ctx);
    kernel::TypeA2B<float, float16>(n, y32, y, ctx);
    ctx->FinishDeviceCompution();  // Sync the stream
    ctx->Delete(y32);
}

template <> void RandomNormal<float16, CUDAContext>(
    const int               n,
    const float             mu,
    const float             sigma,
    float16*                y,
    CUDAContext*            ctx) {
    auto* y32 = (float*)ctx->New(n * sizeof(float));
    math::RandomNormal<float, CUDAContext>(n, mu, sigma, y32, ctx);
    kernel::TypeA2B<float, float16>(n, y32, y, ctx);
    ctx->FinishDeviceCompution();  // Sync the stream
    ctx->Delete(y32);
}

}  // namespace math

}  // namespace dragon

#endif  // WITH_CUDA