
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename IndexT, typename ValueT>
__global__ void
_BooleanMask(const int N, const IndexT* index, const ValueT* x, ValueT* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = x[index[i]];
  }
}

template <typename IndexT, typename ValueT>
__global__ void _BooleanMaskGrad(
    const int N,
    const IndexT* index,
    const ValueT* dy,
    ValueT* dx) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dx[index[i]] = dy[i];
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(IndexT, ValueT)                             \
  template <>                                                              \
  void BooleanMask<IndexT, ValueT, CUDAContext>(                           \
      const int N,                                                         \
      const IndexT* index,                                                 \
      const ValueT* x,                                                     \
      ValueT* y,                                                           \
      CUDAContext* ctx) {                                                  \
    _BooleanMask<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        N, index, x, y);                                                   \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(IndexT, ValueT)                            \
  template <>                                                                  \
  void BooleanMaskGrad<IndexT, ValueT, CUDAContext>(                           \
      const int N,                                                             \
      const IndexT* index,                                                     \
      const ValueT* dy,                                                        \
      ValueT* dx,                                                              \
      CUDAContext* ctx) {                                                      \
    _BooleanMaskGrad<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        N, index, dy, dx);                                                     \
  }

DEFINE_KERNEL_LAUNCHER(int, bool);
DEFINE_KERNEL_LAUNCHER(int, uint8_t);
DEFINE_KERNEL_LAUNCHER(int, int8_t);
DEFINE_KERNEL_LAUNCHER(int, int);
DEFINE_KERNEL_LAUNCHER(int, int64_t);
DEFINE_KERNEL_LAUNCHER(int, float16);
DEFINE_KERNEL_LAUNCHER(int, float);
DEFINE_KERNEL_LAUNCHER(int, double);
DEFINE_KERNEL_LAUNCHER(int64_t, bool);
DEFINE_KERNEL_LAUNCHER(int64_t, uint8_t);
DEFINE_KERNEL_LAUNCHER(int64_t, int8_t);
DEFINE_KERNEL_LAUNCHER(int64_t, int);
DEFINE_KERNEL_LAUNCHER(int64_t, int64_t);
DEFINE_KERNEL_LAUNCHER(int64_t, float16);
DEFINE_KERNEL_LAUNCHER(int64_t, float);
DEFINE_KERNEL_LAUNCHER(int64_t, double);
DEFINE_GRAD_KERNEL_LAUNCHER(int, float16);
DEFINE_GRAD_KERNEL_LAUNCHER(int, float);
DEFINE_GRAD_KERNEL_LAUNCHER(int, double);
DEFINE_GRAD_KERNEL_LAUNCHER(int64_t, float16);
DEFINE_GRAD_KERNEL_LAUNCHER(int64_t, float);
DEFINE_GRAD_KERNEL_LAUNCHER(int64_t, double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
