
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/cast.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T, int D>
__global__ void _ConstPad(
    const int nthreads,
    const int num_dims,
    const SimpleArray<int, D> x_dims,
    const SimpleArray<int, D> x_strides,
    const SimpleArray<int, D> y_dims,
    const SimpleArray<int, D> pads,
    const T value,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    int xi = 0, tmp = yi, d;
    for (d = num_dims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(y_dims.data[d], tmp, &tmp, &r);
      r -= pads.data[d];
      if (r < 0 || r >= x_dims.data[d]) break;
      xi += r * x_strides.data[d];
    }
    y[yi] = d >= 0 ? value : x[xi];
  }
}

template <typename T, int D>
__global__ void _ReflectPad(
    const int nthreads,
    const int num_dims,
    const SimpleArray<int, D> x_dims,
    const SimpleArray<int, D> x_strides,
    const SimpleArray<int, D> y_dims,
    const SimpleArray<int, D> pads,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    int xi = 0, tmp = yi;
    for (int d = num_dims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(y_dims.data[d], tmp, &tmp, &r);
      r -= pads.data[d];
      r = max(r, -r);
      r = min(r, 2 * x_dims.data[d] - r - 2);
      xi += r * x_strides.data[d];
    }
    y[yi] = x[xi];
  }
}

template <typename T, int D>
__global__ void _EdgePad(
    const int nthreads,
    const int num_dims,
    const SimpleArray<int, D> x_dims,
    const SimpleArray<int, D> x_strides,
    const SimpleArray<int, D> y_dims,
    const SimpleArray<int, D> pads,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    int xi = 0, tmp = yi;
    for (int d = num_dims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(y_dims.data[d], tmp, &tmp, &r);
      r = min(x_dims.data[d] - 1, max(r - pads.data[d], 0));
      xi += r * x_strides.data[d];
    }
    y[yi] = x[xi];
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_CONST_KERNEL_LAUNCHER(T)                                        \
  template <>                                                                  \
  void ConstPad<T, CUDAContext>(                                               \
      const int num_dims,                                                      \
      const int64_t* x_dims,                                                   \
      const int64_t* x_strides,                                                \
      const int64_t* y_dims,                                                   \
      const int64_t* pads,                                                     \
      const float value,                                                       \
      const T* x,                                                              \
      T* y,                                                                    \
      CUDAContext* ctx) {                                                      \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                          \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_dims, X_strides, Y_dims, X_pads;  \
    const auto nthreads = std::accumulate(                                     \
        y_dims, y_dims + num_dims, 1, std::multiplies<int64_t>());             \
    for (int i = 0; i < num_dims; ++i) {                                       \
      X_dims.data[i] = x_dims[i];                                              \
      X_strides.data[i] = x_strides[i];                                        \
      Y_dims.data[i] = y_dims[i];                                              \
      X_pads.data[i] = pads[i];                                                \
    }                                                                          \
    _ConstPad<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        nthreads,                                                              \
        num_dims,                                                              \
        X_dims,                                                                \
        X_strides,                                                             \
        Y_dims,                                                                \
        X_pads,                                                                \
        cast::to<T>(value),                                                    \
        x,                                                                     \
        y);                                                                    \
  }

#define DEFINE_KERNEL_LAUNCHER(name, T)                                       \
  template <>                                                                 \
  void name<T, CUDAContext>(                                                  \
      const int num_dims,                                                     \
      const int64_t* x_dims,                                                  \
      const int64_t* x_strides,                                               \
      const int64_t* y_dims,                                                  \
      const int64_t* pads,                                                    \
      const T* x,                                                             \
      T* y,                                                                   \
      CUDAContext* ctx) {                                                     \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                         \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_dims, X_strides, Y_dims, X_pads; \
    const auto nthreads = std::accumulate(                                    \
        y_dims, y_dims + num_dims, 1, std::multiplies<int64_t>());            \
    for (int i = 0; i < num_dims; ++i) {                                      \
      X_dims.data[i] = x_dims[i];                                             \
      X_strides.data[i] = x_strides[i];                                       \
      Y_dims.data[i] = y_dims[i];                                             \
      X_pads.data[i] = pads[i];                                               \
    }                                                                         \
    _##name<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
        nthreads, num_dims, X_dims, X_strides, Y_dims, X_pads, x, y);         \
  }

DEFINE_CONST_KERNEL_LAUNCHER(bool);
DEFINE_CONST_KERNEL_LAUNCHER(int8_t);
DEFINE_CONST_KERNEL_LAUNCHER(uint8_t);
DEFINE_CONST_KERNEL_LAUNCHER(int);
DEFINE_CONST_KERNEL_LAUNCHER(int64_t);
DEFINE_CONST_KERNEL_LAUNCHER(float16);
DEFINE_CONST_KERNEL_LAUNCHER(float);
DEFINE_CONST_KERNEL_LAUNCHER(double);

DEFINE_KERNEL_LAUNCHER(ReflectPad, bool);
DEFINE_KERNEL_LAUNCHER(ReflectPad, int8_t);
DEFINE_KERNEL_LAUNCHER(ReflectPad, uint8_t);
DEFINE_KERNEL_LAUNCHER(ReflectPad, int);
DEFINE_KERNEL_LAUNCHER(ReflectPad, int64_t);
DEFINE_KERNEL_LAUNCHER(ReflectPad, float16);
DEFINE_KERNEL_LAUNCHER(ReflectPad, float);
DEFINE_KERNEL_LAUNCHER(ReflectPad, double);

DEFINE_KERNEL_LAUNCHER(EdgePad, bool);
DEFINE_KERNEL_LAUNCHER(EdgePad, int8_t);
DEFINE_KERNEL_LAUNCHER(EdgePad, uint8_t);
DEFINE_KERNEL_LAUNCHER(EdgePad, int);
DEFINE_KERNEL_LAUNCHER(EdgePad, int64_t);
DEFINE_KERNEL_LAUNCHER(EdgePad, float16);
DEFINE_KERNEL_LAUNCHER(EdgePad, float);
DEFINE_KERNEL_LAUNCHER(EdgePad, double);

#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_CONST_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
