
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T, int D>
__global__ void _Transpose(
    const int nthreads,
    const int ndims,
    const SimpleArray<int, D> x_strides,
    const SimpleArray<int, D> y_dims,
    const T* x,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    int xi = 0, tmp = yi;
    for (int d = ndims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(y_dims.data[d], tmp, &tmp, &r);
      xi += r * x_strides.data[d];
    }
    y[yi] = x[xi];
  }
}

template <typename T, int D>
__global__ void _TransposeGrad(
    const int nthreads,
    const int ndims,
    const SimpleArray<int, D> x_strides,
    const SimpleArray<int, D> y_dims,
    const T* dy,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    int xi = 0, tmp = yi;
    for (int d = ndims - 1; d >= 0; --d) {
      int r;
      FIXED_DIVISOR_DIV_MOD(y_dims.data[d], tmp, &tmp, &r);
      xi += r * x_strides.data[d];
    }
    dx[xi] = dy[yi];
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(name, T)                                      \
  template <>                                                                \
  void name<T, CUDAContext>(                                                 \
      const int num_dims,                                                    \
      const int64_t* x_strides,                                              \
      const int64_t* y_dims,                                                 \
      const T* x,                                                            \
      T* y,                                                                  \
      CUDAContext* ctx) {                                                    \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                        \
    SimpleArray<int, CUDA_TENSOR_MAX_DIMS> X_strides, Y_dims;                \
    const auto nthreads = std::accumulate(                                   \
        y_dims, y_dims + num_dims, 1, std::multiplies<int64_t>());           \
    for (int i = 0; i < num_dims; ++i) {                                     \
      X_strides.data[i] = x_strides[i];                                      \
      Y_dims.data[i] = y_dims[i];                                            \
    }                                                                        \
    _##name<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        nthreads, num_dims, X_strides, Y_dims, x, y);                        \
  }

DEFINE_KERNEL_LAUNCHER(Transpose, bool);
DEFINE_KERNEL_LAUNCHER(Transpose, int8_t);
DEFINE_KERNEL_LAUNCHER(Transpose, uint8_t);
DEFINE_KERNEL_LAUNCHER(Transpose, int);
DEFINE_KERNEL_LAUNCHER(Transpose, int64_t);
DEFINE_KERNEL_LAUNCHER(Transpose, float16);
DEFINE_KERNEL_LAUNCHER(Transpose, float);
DEFINE_KERNEL_LAUNCHER(Transpose, double);

DEFINE_KERNEL_LAUNCHER(TransposeGrad, float16);
DEFINE_KERNEL_LAUNCHER(TransposeGrad, float);
DEFINE_KERNEL_LAUNCHER(TransposeGrad, double);

#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
