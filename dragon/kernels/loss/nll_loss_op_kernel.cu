
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename LogitType, typename TargetType>
__global__ void _NLLLoss(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const int ignore_index,
    const LogitType* logit,
    const TargetType* target,
    LogitType* loss,
    LogitType* mask) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int i = yi / inner_dim;
    const int j = yi % inner_dim;
    const int label = target[i * inner_dim + j];
    if (label == ignore_index) {
      loss[yi] = mask[yi] = LogitType(0);
    } else {
      loss[yi] = -logit[(i * axis_dim + label) * inner_dim + j];
      mask[yi] = LogitType(1);
    }
  }
}

template <typename LogitType, typename TargetType>
__global__ void _NLLLossGrad(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const int ignore_index,
    const LogitType* logit,
    const TargetType* target,
    LogitType* dlogit,
    LogitType* mask) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int i = yi / inner_dim;
    const int j = yi % inner_dim;
    const int label = target[i * inner_dim + j];
    if (label == ignore_index) {
      mask[yi] = LogitType(0);
    } else {
      dlogit[(i * axis_dim + label) * inner_dim + j] = LogitType(-1);
      mask[yi] = LogitType(1);
    }
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(name, LogitType, TargetType)                  \
  template <>                                                                \
  void name<LogitType, TargetType, CUDAContext>(                             \
      const int outer_dim,                                                   \
      const int inner_dim,                                                   \
      const int axis_dim,                                                    \
      const int ignore_index,                                                \
      const LogitType* logit,                                                \
      const TargetType* target,                                              \
      LogitType* loss,                                                       \
      LogitType* mask,                                                       \
      CUDAContext* ctx) {                                                    \
    const auto nthreads = outer_dim * inner_dim;                             \
    _##name<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        nthreads,                                                            \
        inner_dim,                                                           \
        axis_dim,                                                            \
        ignore_index,                                                        \
        logit,                                                               \
        target,                                                              \
        loss,                                                                \
        mask);                                                               \
  }

DEFINE_KERNEL_LAUNCHER(NLLLoss, float, float);
DEFINE_KERNEL_LAUNCHER(NLLLoss, float, int64_t);
DEFINE_KERNEL_LAUNCHER(NLLLoss, double, double);
DEFINE_KERNEL_LAUNCHER(NLLLoss, double, int64_t);

DEFINE_KERNEL_LAUNCHER(NLLLossGrad, float, float);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, float, int64_t);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, double, double);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, double, int64_t);

#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
