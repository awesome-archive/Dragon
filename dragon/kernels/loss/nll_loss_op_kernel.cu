
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename LogitT, typename TargetT>
__global__ void _NLLLoss(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const int ignore_index,
    const LogitT* logit,
    const TargetT* target,
    LogitT* loss,
    LogitT* mask) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int i = yi / inner_dim;
    const int j = yi % inner_dim;
    const int label = target[i * inner_dim + j];
    if (label == ignore_index) {
      loss[yi] = mask[yi] = LogitT(0);
    } else {
      loss[yi] = -logit[(i * axis_dim + label) * inner_dim + j];
      mask[yi] = LogitT(1);
    }
  }
}

template <typename LogitT, typename TargetT>
__global__ void _NLLLossGrad(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const int ignore_index,
    const LogitT* logit,
    const TargetT* target,
    LogitT* dlogit,
    LogitT* mask) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int i = yi / inner_dim;
    const int j = yi % inner_dim;
    const int label = target[i * inner_dim + j];
    if (label == ignore_index) {
      mask[yi] = LogitT(0);
    } else {
      dlogit[(i * axis_dim + label) * inner_dim + j] = LogitT(-1);
      mask[yi] = LogitT(1);
    }
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(name, LogitT, TargetT)                        \
  template <>                                                                \
  void name<LogitT, TargetT, CUDAContext>(                                   \
      const int outer_dim,                                                   \
      const int inner_dim,                                                   \
      const int axis_dim,                                                    \
      const int ignore_index,                                                \
      const LogitT* logit,                                                   \
      const TargetT* target,                                                 \
      LogitT* loss,                                                          \
      LogitT* mask,                                                          \
      CUDAContext* ctx) {                                                    \
    const auto nthreads = outer_dim * inner_dim;                             \
    _##name<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        nthreads,                                                            \
        inner_dim,                                                           \
        axis_dim,                                                            \
        ignore_index,                                                        \
        logit,                                                               \
        target,                                                              \
        loss,                                                                \
        mask);                                                               \
  }

DEFINE_KERNEL_LAUNCHER(NLLLoss, float, float);
DEFINE_KERNEL_LAUNCHER(NLLLoss, float, int64_t);
DEFINE_KERNEL_LAUNCHER(NLLLoss, double, double);
DEFINE_KERNEL_LAUNCHER(NLLLoss, double, int64_t);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, float, float);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, float, int64_t);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, double, double);
DEFINE_KERNEL_LAUNCHER(NLLLossGrad, double, int64_t);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
