
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename LogitT, typename TargetT>
__global__ void _SigmoidFocalLoss(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const LogitT pos_alpha,
    const LogitT neg_alpha,
    const LogitT gamma,
    const int negative_index,
    const LogitT* logit,
    const TargetT* target,
    LogitT* loss,
    LogitT* mask) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int j = yi % inner_dim;
    const int k = (yi / inner_dim) % axis_dim;
    const int i = yi / inner_dim / axis_dim;
    const int t = target[i * inner_dim + j];

    // "0" is reserved for target if negative index is zero
    LogitT c1 = (LogitT)(t == (k + (negative_index ? 0 : 1)));
    LogitT c2 = (LogitT)((t >= 0) & (t != (k + (negative_index ? 0 : 1))));
    LogitT p = LogitT(1) / (LogitT(1) + exp(-logit[yi]));

    // (1 - p)^{gamma} * log(p)
    LogitT pos_term = pow(LogitT(1) - p, gamma) * log(max(p, FLT_MIN));

    // p^{gamma} * log(1 - p)
    LogitT neg_term = pow(p, gamma) *
        (-logit[yi] * (logit[yi] >= 0) -
         log(LogitT(1) +
             exp(logit[yi] - LogitT(2) * logit[yi] * (logit[yi] >= 0))));

    loss[yi] = LogitT(0);
    loss[yi] += -c1 * pos_term * pos_alpha;
    loss[yi] += -c2 * neg_term * neg_alpha;
    mask[yi] = c1;
  }
}

template <typename LogitT, typename TargetT>
__global__ void _SigmoidFocalLossGrad(
    const int nthreads,
    const int inner_dim,
    const int axis_dim,
    const LogitT pos_alpha,
    const LogitT neg_alpha,
    const LogitT gamma,
    const int negative_index,
    const LogitT* logit,
    const TargetT* target,
    LogitT* dx,
    LogitT* mask) {
  CUDA_1D_KERNEL_LOOP(xi, nthreads) {
    const int j = xi % inner_dim;
    const int k = (xi / inner_dim) % axis_dim;
    const int i = xi / inner_dim / axis_dim;
    const int t = target[i * inner_dim + j];

    // "0" is reserved for target if neg index is zero
    LogitT c1 = (LogitT)(t == (k + (negative_index ? 0 : 1)));
    LogitT c2 = (LogitT)((t >= 0) & (t != (k + (negative_index ? 0 : 1))));
    LogitT p = LogitT(1) / (LogitT(1) + exp(-logit[xi]));

    // (1 - p)^{gamma} * (1 - p - gamma * p * log(p))
    LogitT pos_term = pow(LogitT(1) - p, gamma) *
        (LogitT(1) - p - p * gamma * log(max(p, FLT_MIN)));

    // p^{gamma} * (gamma * (1 - p) * log(1-p) - p)
    LogitT neg_term = pow(p, gamma) *
        ((-logit[xi] * (logit[xi] >= 0) -
          log(LogitT(1) +
              exp(logit[xi] - LogitT(2) * logit[xi] * (logit[xi] >= 0)))) *
             (LogitT(1) - p) * gamma -
         p);

    dx[xi] = LogitT(0);
    dx[xi] += -c1 * pos_term * pos_alpha;
    dx[xi] += -c2 * neg_term * neg_alpha;
    mask[xi] = c1;
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(name, LogitT, TargetT)                        \
  template <>                                                                \
  void name<LogitT, TargetT, CUDAContext>(                                   \
      const int outer_dim,                                                   \
      const int inner_dim,                                                   \
      const int axis_dim,                                                    \
      const float pos_alpha,                                                 \
      const float neg_alpha,                                                 \
      const float gamma,                                                     \
      const int negative_index,                                              \
      const LogitT* logit,                                                   \
      const TargetT* target,                                                 \
      LogitT* loss,                                                          \
      LogitT* mask,                                                          \
      CUDAContext* ctx) {                                                    \
    const auto nthreads = outer_dim * axis_dim * inner_dim;                  \
    _##name<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        nthreads,                                                            \
        inner_dim,                                                           \
        axis_dim,                                                            \
        (LogitT)pos_alpha,                                                   \
        (LogitT)neg_alpha,                                                   \
        (LogitT)gamma,                                                       \
        negative_index,                                                      \
        logit,                                                               \
        target,                                                              \
        loss,                                                                \
        mask);                                                               \
  }

DEFINE_KERNEL_LAUNCHER(SigmoidFocalLoss, float, float);
DEFINE_KERNEL_LAUNCHER(SigmoidFocalLoss, float, int64_t);
DEFINE_KERNEL_LAUNCHER(SigmoidFocalLoss, double, double);
DEFINE_KERNEL_LAUNCHER(SigmoidFocalLoss, double, int64_t);
DEFINE_KERNEL_LAUNCHER(SigmoidFocalLossGrad, float, float);
DEFINE_KERNEL_LAUNCHER(SigmoidFocalLossGrad, float, int64_t);
DEFINE_KERNEL_LAUNCHER(SigmoidFocalLossGrad, double, double);
DEFINE_KERNEL_LAUNCHER(SigmoidFocalLossGrad, double, int64_t);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
