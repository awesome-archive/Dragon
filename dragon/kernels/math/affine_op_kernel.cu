
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T>
__global__ void _Affine(
    const int nthreads,
    const int axis_dim,
    const int inner_dim,
    const T* x,
    const T* w,
    T* y) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 350
    y[i] = __ldg(w + (i / inner_dim) % axis_dim) * x[i];
#else
    y[i] = w[(i / inner_dim) % axis_dim] * x[i];
#endif
  }
}

template <>
__global__ void _Affine<half>(
    const int nthreads,
    const int axis_dim,
    const int inner_dim,
    const half* x,
    const half* w,
    half* y) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    y[i] = __hmul(x[i], __ldg(w + (i / inner_dim) % axis_dim));
#endif
  }
}

template <typename T>
__global__ void _Affine(
    const int nthreads,
    const int axis_dim,
    const int inner_dim,
    const T* x,
    const T* w,
    const T* b,
    T* y) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const int wi = (i / inner_dim) % axis_dim;
#if __CUDA_ARCH__ >= 350
    y[i] = __ldg(w + wi) * x[i] + __ldg(b + wi);
#else
    y[i] = w[wi] * x[i] + b[wi];
#endif
  }
}

template <>
__global__ void _Affine<half>(
    const int nthreads,
    const int axis_dim,
    const int inner_dim,
    const half* x,
    const half* w,
    const half* b,
    half* y) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    const int wi = (i / inner_dim) % axis_dim;
    y[i] = __hadd(__hmul(x[i], __ldg(w + wi)), __ldg(b + wi));
#endif
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void Affine<float16, CUDAContext>(
    const int outer_dim,
    const int axis_dim,
    const int inner_dim,
    const float16* x,
    const float16* w,
    const float16* b,
    float16* y,
    CUDAContext* ctx) {
  const int nthreads = outer_dim * axis_dim * inner_dim;
  if (b != nullptr) {
    _Affine<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
        nthreads,
        axis_dim,
        inner_dim,
        reinterpret_cast<const half*>(x),
        reinterpret_cast<const half*>(w),
        reinterpret_cast<const half*>(b),
        reinterpret_cast<half*>(y));
  } else {
    _Affine<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
        nthreads,
        axis_dim,
        inner_dim,
        reinterpret_cast<const half*>(x),
        reinterpret_cast<const half*>(w),
        reinterpret_cast<half*>(y));
  }
}

#define DEFINE_KERNEL_LAUNCHER(T)                                              \
  template <>                                                                  \
  void Affine<T, CUDAContext>(                                                 \
      const int outer_dim,                                                     \
      const int axis_dim,                                                      \
      const int inner_dim,                                                     \
      const T* x,                                                              \
      const T* w,                                                              \
      const T* b,                                                              \
      T* y,                                                                    \
      CUDAContext* ctx) {                                                      \
    const int nthreads = outer_dim * axis_dim * inner_dim;                     \
    if (b != nullptr) {                                                        \
      _Affine<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
          nthreads, axis_dim, inner_dim, x, w, b, y);                          \
    } else {                                                                   \
      _Affine<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
          nthreads, axis_dim, inner_dim, x, w, y);                             \
    }                                                                          \
  }

DEFINE_KERNEL_LAUNCHER(int8_t);
DEFINE_KERNEL_LAUNCHER(uint8_t);
DEFINE_KERNEL_LAUNCHER(int);
DEFINE_KERNEL_LAUNCHER(int64_t);
DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);

#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
