
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/conversions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T, typename AccT>
__global__ void
_Clip(const int nthreads, const AccT low, const AccT high, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    y[i] = convert::To<T>(max(low, min(convert::To<AccT>(x[i]), high)));
  }
}

template <typename T, typename AccT>
__global__ void _ClipGrad(
    const int nthreads,
    const AccT low,
    const AccT high,
    const T* dy,
    const T* x,
    T* dx) {
  const T kZero = convert::To<T>(0.f);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const AccT val = convert::To<AccT>(x[i]);
    dx[i] = val < low || val > high ? kZero : dy[i];
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(T, AccT)                                \
  template <>                                                          \
  void Clip<T, CUDAContext>(                                           \
      const int count,                                                 \
      const float low,                                                 \
      const float high,                                                \
      const T* x,                                                      \
      T* y,                                                            \
      CUDAContext* ctx) {                                              \
    _Clip<T, AccT>                                                     \
        <<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
            count, low, high, x, y);                                   \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T, AccT)                           \
  template <>                                                          \
  void ClipGrad<T, CUDAContext>(                                       \
      const int count,                                                 \
      const float low,                                                 \
      const float high,                                                \
      const T* dy,                                                     \
      const T* x,                                                      \
      T* dx,                                                           \
      CUDAContext* ctx) {                                              \
    _ClipGrad<T, AccT>                                                 \
        <<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
            count, low, high, dy, x, dx);                              \
  }

DEFINE_KERNEL_LAUNCHER(int8_t, int8_t);
DEFINE_KERNEL_LAUNCHER(uint8_t, uint8_t);
DEFINE_KERNEL_LAUNCHER(int, int);
DEFINE_KERNEL_LAUNCHER(int64_t, int64_t);
DEFINE_KERNEL_LAUNCHER(float16, float);
DEFINE_KERNEL_LAUNCHER(float, float);
DEFINE_KERNEL_LAUNCHER(double, double);
DEFINE_GRAD_KERNEL_LAUNCHER(float16, float);
DEFINE_GRAD_KERNEL_LAUNCHER(float, float);
DEFINE_GRAD_KERNEL_LAUNCHER(double, double);
#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
