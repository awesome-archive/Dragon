
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T>
__global__ void
_Clip(const int nthreads, const T low, const T high, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    y[i] = max(low, min(x[i], high));
  }
}

template <>
__global__ void _Clip<half>(
    const int nthreads,
    const half low,
    const half high,
    const half* x,
    half* y) {
#if __CUDA_ARCH__ >= 530
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    y[i] = __hlt(__ldg(x + i), high)
        ? (__hgt(__ldg(x + i), low) ? __ldg(x + i) : low)
        : high;
  }
#else
  const float kLow = __half2float(low);
  const float kHigh = __half2float(high);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    y[i] = __float2half(max(kLow, min(__half2float(x[i]), kHigh)));
  }
#endif
}

template <typename T>
__global__ void _ClipGrad(
    const int nthreads,
    const T low,
    const T high,
    const T* dy,
    const T* x,
    T* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 350
    dx[i] = __ldg(x + i) < low || __ldg(x + i) > high ? T(0) : dy[i];
#else
    dx[i] = x[i] < low || x[i] > high ? T(0) : dy[i];
#endif
  }
}

template <>
__global__ void _ClipGrad<half>(
    const int nthreads,
    const half low,
    const half high,
    const half* dy,
    const half* x,
    half* dx) {
  const half kZero = __float2half(0.f);
#if __CUDA_ARCH__ >= 530
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    dx[i] =
        (__hlt(__ldg(x + i), low) || __hgt(__ldg(x + i), high)) ? kZero : dy[i];
  }
#elif __CUDA_ARCH__ >= 350
  const float kLow = __half2float(low);
  const float kHigh = __half2float(high);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    dx[i] = (__half2float(__ldg(x + i)) < kLow ||
             __half2float(__ldg(x + i)) > kHigh)
        ? kZero
        : dy[i];
  }
#else
  const float kLow = __half2float(low);
  const float kHigh = __half2float(high);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    dx[i] = (__half2float(x[i]) < kLow || __half2float(x[i]) > kHigh) ? kZero
                                                                      : dy[i];
  }
#endif
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void Clip<float16, CUDAContext>(
    const int count,
    const float low,
    const float high,
    const float16* x,
    float16* y,
    CUDAContext* ctx) {
  _Clip<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      count,
      convert::To<half>(low),
      convert::To<half>(high),
      reinterpret_cast<const half*>(x),
      reinterpret_cast<half*>(y));
}

template <>
void ClipGrad<float16, CUDAContext>(
    const int count,
    const float low,
    const float high,
    const float16* dy,
    const float16* x,
    float16* dx,
    CUDAContext* ctx) {
  _ClipGrad<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      count,
      convert::To<half>(low),
      convert::To<half>(high),
      reinterpret_cast<const half*>(dy),
      reinterpret_cast<const half*>(x),
      reinterpret_cast<half*>(dx));
} // ClipGrad

#define DEFINE_KERNEL_LAUNCHER(T)                                       \
  template <>                                                           \
  void Clip<T, CUDAContext>(                                            \
      const int count,                                                  \
      const float low,                                                  \
      const float high,                                                 \
      const T* x,                                                       \
      T* y,                                                             \
      CUDAContext* ctx) {                                               \
    _Clip<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        count, convert::To<T>(low), convert::To<T>(high), x, y);        \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                      \
  template <>                                                               \
  void ClipGrad<T, CUDAContext>(                                            \
      const int count,                                                      \
      const float low,                                                      \
      const float high,                                                     \
      const T* dy,                                                          \
      const T* x,                                                           \
      T* dx,                                                                \
      CUDAContext* ctx) {                                                   \
    _ClipGrad<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        count, convert::To<T>(low), convert::To<T>(high), dy, x, dx);       \
  }

DEFINE_KERNEL_LAUNCHER(int8_t);
DEFINE_KERNEL_LAUNCHER(uint8_t);
DEFINE_KERNEL_LAUNCHER(int);
DEFINE_KERNEL_LAUNCHER(int64_t);
DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);

DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);

#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
