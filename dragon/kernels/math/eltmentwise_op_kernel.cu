
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T>
__global__ void _CosGrad(const int nthreads, const T* dy, const T* x, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    dx[i] = -dy[i] * sin(x[i]);
  }
}

template <>
__global__ void
_CosGrad<half>(const int nthreads, const half* dy, const half* x, half* dx) {
  const half kFactor = __float2half(-1.f);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] = __hmul(__hmul(dy[i], kFactor), hsin(x[i]));
#endif
  }
}

template <>
__global__ void _CosGrad<half2>(
    const int nthreads,
    const half2* dy,
    const half2* x,
    half2* dx) {
  const half2 kFactor = __float2half2_rn(-1.f);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] = __hmul2(__hmul2(dy[i], kFactor), h2sin(x[i]));
#endif
  }
}

template <typename T>
__global__ void _SinGrad(const int nthreads, const T* dy, const T* x, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    dx[i] = dy[i] * cos(x[i]);
  }
}

template <>
__global__ void
_SinGrad<half>(const int nthreads, const half* dy, const half* x, half* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] = __hmul(dy[i], hcos(x[i]));
#endif
  }
}

template <>
__global__ void _SinGrad<half2>(
    const int nthreads,
    const half2* dy,
    const half2* x,
    half2* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] = __hmul2(dy[i], h2cos(x[i]));
#endif
  }
}

template <typename T>
__global__ void
_ReciprocalGrad(const int nthreads, const T* dy, const T* y, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    dx[i] = -dy[i] * utils::math::Square(y[i]);
  }
}

template <>
__global__ void _ReciprocalGrad<half>(
    const int nthreads,
    const half* dy,
    const half* y,
    half* dx) {
  const half c = __float2half(-1.f);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] = __hmul(__hmul(c, dy[i]), utils::math::Square(y[i]));
#endif
  }
}

template <>
__global__ void _ReciprocalGrad<half2>(
    const int nthreads,
    const half2* dy,
    const half2* y,
    half2* dx) {
  const half2 c = __float2half2_rn(-1.f);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] = __hmul2(__hmul2(c, dy[i]), utils::math::Square(y[i]));
#endif
  }
}

template <typename T>
__global__ void _RsqrtGrad(const int nthreads, const T* dy, const T* y, T* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    dx[i] = T(-0.5) * dy[i] * utils::math::Cube(y[i]);
  }
}

template <>
__global__ void
_RsqrtGrad<half>(const int nthreads, const half* dy, const half* y, half* dx) {
  const half c = __float2half(-0.5f);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] = __hmul(__hmul(c, dy[i]), utils::math::Cube(y[i]));
#endif
  }
}

template <>
__global__ void _RsqrtGrad<half2>(
    const int nthreads,
    const half2* dy,
    const half2* y,
    half2* dx) {
  const half2 c = __float2half2_rn(-0.5f);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] = __hmul2(__hmul2(c, dy[i]), utils::math::Cube(y[i]));
#endif
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_GRAD_KERNEL_LAUNCHER(name, T)                               \
  template <>                                                              \
  void name##Grad<T, CUDAContext>(                                         \
      const int count, const T* dy, const T* x, T* dx, CUDAContext* ctx) { \
    _##name##Grad<<<                                                       \
        CUDA_BLOCKS(count),                                                \
        CUDA_THREADS,                                                      \
        0,                                                                 \
        ctx->cuda_stream()>>>(count, dy, x, dx);                           \
  }

DEFINE_GRAD_KERNEL_LAUNCHER(Cos, float);
DEFINE_GRAD_KERNEL_LAUNCHER(Cos, double);
DEFINE_GRAD_KERNEL_LAUNCHER(Sin, float);
DEFINE_GRAD_KERNEL_LAUNCHER(Sin, double);
DEFINE_GRAD_KERNEL_LAUNCHER(Reciprocal, float);
DEFINE_GRAD_KERNEL_LAUNCHER(Reciprocal, double);
DEFINE_GRAD_KERNEL_LAUNCHER(Rsqrt, float);
DEFINE_GRAD_KERNEL_LAUNCHER(Rsqrt, double);
#undef DEFINE_GRAD_KERNEL_LAUNCHER

#define DEFINE_GRAD_KERNEL_LAUNCHER(name)     \
  template <>                                 \
  void name##Grad<float16, CUDAContext>(      \
      const int count,                        \
      const float16* dy,                      \
      const float16* x,                       \
      float16* dx,                            \
      CUDAContext* ctx) {                     \
    if ((count & 1) == 0) {                   \
      _##name##Grad<<<                        \
          CUDA_BLOCKS(count >> 1),            \
          CUDA_THREADS,                       \
          0,                                  \
          ctx->cuda_stream()>>>(              \
          count >> 1,                         \
          reinterpret_cast<const half2*>(dy), \
          reinterpret_cast<const half2*>(x),  \
          reinterpret_cast<half2*>(dx));      \
    } else {                                  \
      _##name##Grad<<<                        \
          CUDA_BLOCKS(count),                 \
          CUDA_THREADS,                       \
          0,                                  \
          ctx->cuda_stream()>>>(              \
          count,                              \
          reinterpret_cast<const half*>(dy),  \
          reinterpret_cast<const half*>(x),   \
          reinterpret_cast<half*>(dx));       \
    }                                         \
  }

DEFINE_GRAD_KERNEL_LAUNCHER(Cos);
DEFINE_GRAD_KERNEL_LAUNCHER(Sin);
DEFINE_GRAD_KERNEL_LAUNCHER(Reciprocal);
DEFINE_GRAD_KERNEL_LAUNCHER(Rsqrt);
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
