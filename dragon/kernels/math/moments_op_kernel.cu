
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/device/common_cub.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T, typename AccT>
__global__ void _RowwiseMoments(
    const int rows,
    const int cols,
    const T* x,
    AccT* mean,
    AccT* var) {
  __shared__ typename BlockReduce<AccT>::TempStorage m_storage;
  __shared__ typename BlockReduce<AccT>::TempStorage v_storage;
  const AccT scale = AccT(1) / AccT(rows);
  CUDA_2D_KERNEL_LOOP1(i, cols) {
    AccT m_val = AccT(0), v_val = AccT(0);
    CUDA_2D_KERNEL_LOOP2(j, rows) {
      const AccT val = convert::To<AccT>(x[j * cols + i]);
      m_val += val;
      v_val += val * val;
    }
    m_val = BlockReduce<AccT>(m_storage).Sum(m_val);
    v_val = BlockReduce<AccT>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      mean[i] = m_val = m_val * scale;
      var[i] = v_val * scale - m_val * m_val;
    }
  }
}

template <typename T, typename AccT>
__global__ void _ColwiseMoments(
    const int rows,
    const int cols,
    const T* x,
    AccT* mean,
    AccT* var) {
  __shared__ typename BlockReduce<AccT>::TempStorage m_storage;
  __shared__ typename BlockReduce<AccT>::TempStorage v_storage;
  const AccT scale = AccT(1) / AccT(cols);
  CUDA_2D_KERNEL_LOOP1(i, rows) {
    AccT m_val = AccT(0), v_val = AccT(0);
    CUDA_2D_KERNEL_LOOP2(j, cols) {
      const AccT val = convert::To<AccT>(x[i * cols + j]);
      m_val += val;
      v_val += val * val;
    }
    m_val = BlockReduce<AccT>(m_storage).Sum(m_val);
    v_val = BlockReduce<AccT>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      mean[i] = m_val = m_val * scale;
      var[i] = v_val * scale - m_val * m_val;
    }
  }
}

template <typename T, typename AccT, int D>
__global__ void _GenericMoments(
    const int rows,
    const int cols,
    const SimpleArray<int, D> X_dims,
    const SimpleArray<int, D> X_strides,
    const T* x,
    AccT* mean,
    AccT* var) {
  __shared__ typename BlockReduce<AccT>::TempStorage m_storage;
  __shared__ typename BlockReduce<AccT>::TempStorage v_storage;
  const AccT scale = AccT(1) / AccT(cols);
  CUDA_2D_KERNEL_LOOP1(i, rows) {
    AccT m_val = AccT(0), v_val = AccT(0);
    CUDA_2D_KERNEL_LOOP2(j, cols) {
      int xi = 0, c = i * cols + j;
#pragma unroll
      for (int d = D - 1; d >= 0; --d) {
        int r;
        FIXED_DIVISOR_DIV_MOD(X_dims.data[d], c, &c, &r);
        xi += r * X_strides.data[d];
      }
      const AccT val = convert::To<AccT>(x[xi]);
      m_val += val;
      v_val += val * val;
    }
    m_val = BlockReduce<AccT>(m_storage).Sum(m_val);
    v_val = BlockReduce<AccT>(v_storage).Sum(v_val);
    if (threadIdx.x == 0) {
      mean[i] = m_val = m_val * scale;
      var[i] = v_val * scale - m_val * m_val;
    }
  }
}

template <typename T, typename AccT, int D>
void _GenericMomentsImpl(
    const int* dims,
    const int num_axes,
    const int* axes,
    const T* x,
    AccT* mean,
    AccT* var,
    CUDAContext* ctx) {
  SimpleArray<int, D> transpose_axes;
  SimpleArray<int, D> transpose_strides;
  SimpleArray<int, D> transpose_dims;
  math::utils::TransposeAxesForReduce(D, num_axes, axes, transpose_axes.data);
  math::utils::ComputeTransposeStrides(
      D, dims, transpose_axes.data, transpose_strides.data);
  int rows = 1, cols = 1;
  const int pivot = D - num_axes;
  for (int i = 0; i < pivot; ++i) {
    rows *= dims[transpose_axes.data[i]];
  }
  for (int i = pivot; i < D; ++i) {
    cols *= dims[transpose_axes.data[i]];
  }
  for (int i = 0; i < D; ++i) {
    transpose_dims.data[i] = dims[transpose_axes.data[i]];
  }
  _GenericMoments<<<rows, CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      rows, cols, transpose_dims, transpose_strides, x, mean, var);
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_KERNEL_LAUNCHER(T, AccT)                               \
  template <>                                                         \
  void Moments<T, AccT, CUDAContext>(                                 \
      const int num_dims,                                             \
      const int* dims,                                                \
      const int num_axes,                                             \
      const int* axes,                                                \
      const T* x,                                                     \
      AccT* mean,                                                     \
      AccT* var,                                                      \
      CUDAContext* ctx) {                                             \
    int rows, cols;                                                   \
    vec32_t out_dims(dims, dims + num_dims);                          \
    for (int i = 0; i < num_axes; ++i) {                              \
      out_dims[axes[i]] = 1;                                          \
    }                                                                 \
    if (math::utils::IsRowwiseReduce(                                 \
            num_dims, dims, out_dims.data(), &rows, &cols)) {         \
      _RowwiseMoments<<<cols, CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
          rows, cols, x, mean, var);                                  \
      return;                                                         \
    }                                                                 \
    if (math::utils::IsColwiseReduce(                                 \
            num_dims, dims, out_dims.data(), &rows, &cols)) {         \
      _ColwiseMoments<<<rows, CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
          rows, cols, x, mean, var);                                  \
      return;                                                         \
    }                                                                 \
    CUDA_TENSOR_DIMS_CHECK(num_dims);                                 \
    DISPATCH_FUNC_BY_VALUE_WITH_TYPE_2(                               \
        _GenericMomentsImpl,                                          \
        T,                                                            \
        AccT,                                                         \
        num_dims,                                                     \
        dims,                                                         \
        num_axes,                                                     \
        axes,                                                         \
        x,                                                            \
        mean,                                                         \
        var,                                                          \
        ctx);                                                         \
  }

DEFINE_KERNEL_LAUNCHER(uint8_t, float);
DEFINE_KERNEL_LAUNCHER(int8_t, float);
DEFINE_KERNEL_LAUNCHER(int, float);
DEFINE_KERNEL_LAUNCHER(int64_t, double);
DEFINE_KERNEL_LAUNCHER(float16, float);
DEFINE_KERNEL_LAUNCHER(float, float);
DEFINE_KERNEL_LAUNCHER(double, double);
#undef DEFINE_KERNEL_LAUNCHER

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
