
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/memory.h"
#include "dragon/utils/device/common_cub.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

#if __CUDA_ARCH__ >= 350
#define LOAD(x, i) __ldg(x + i)
#else
#define LOAD(x, i) x[i]
#endif

namespace {

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormExpectation(
    const int N,
    const int C,
    const int S,
    const Tp denorm,
    const Tx* x,
    Tp* ex,
    Tp* ex2) {
  const int outer_dim = N * S;
  __shared__ typename BlockReduce<Tp>::TempStorage ex_storage;
  __shared__ typename BlockReduce<Tp>::TempStorage ex2_storage;
  CUDA_2D_KERNEL_LOOP1(i, C) {
    Tp ex_val = Tp(0), ex2_val = Tp(0);
    CUDA_2D_KERNEL_LOOP2(j, outer_dim) {
      const int xi = kOrder == StorageOrder::NCHW ? (j / S * C + i) * S + j % S
                                                  : j * C + i;
      ex_val += LOAD(x, xi);
      ex2_val += utils::math::Square(LOAD(x, xi));
    }
    ex_val = BlockReduce<Tp>(ex_storage).Reduce(ex_val, cub::Sum());
    ex2_val = BlockReduce<Tp>(ex2_storage).Reduce(ex2_val, cub::Sum());
    if (threadIdx.x == 0) {
      ex[i] = ex_val * denorm;
      ex2[i] = ex2_val * denorm;
    }
  }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormInternalGrad(
    const int N,
    const int C,
    const int S,
    const Tx* x,
    const Tp* mu,
    const Tp* rsig,
    const Tp* gamma,
    const Tx* dy,
    Tp* dgamma,
    Tp* dbeta) {
  const int outer_dim = N * S;
  __shared__ typename BlockReduce<Tp>::TempStorage dg_storage;
  __shared__ typename BlockReduce<Tp>::TempStorage db_storage;
  CUDA_2D_KERNEL_LOOP1(i, C) {
    Tp dg_val = Tp(0), db_val = Tp(0);
    CUDA_2D_KERNEL_LOOP2(j, outer_dim) {
      const int xi = kOrder == StorageOrder::NCHW ? (j / S * C + i) * S + j % S
                                                  : j * C + i;
      dg_val += LOAD(dy, xi) * (LOAD(x, xi) - LOAD(mu, i)) * LOAD(rsig, i);
      db_val += LOAD(dy, xi);
    }
    dg_val = BlockReduce<Tp>(dg_storage).Reduce(dg_val, cub::Sum());
    db_val = BlockReduce<Tp>(db_storage).Reduce(db_val, cub::Sum());
    if (threadIdx.x == 0) {
      dgamma[i] = dg_val;
      dbeta[i] = db_val;
    }
  }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormTrainingGrad(
    const int nthreads,
    const int N,
    const int C,
    const int S,
    const Tx* x,
    const Tp* mu,
    const Tp* rsig,
    const Tp* gamma,
    const Tp* dgamma,
    const Tp* dbeta,
    const Tx* dy,
    Tx* dx) {
  const Tp denom = Tp(1) / Tp(N * S);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const int pi = kOrder == StorageOrder::NCHW ? (i / S) % C : i % C;
    const Tp x_norm = (LOAD(x, i) - LOAD(mu, pi)) * LOAD(rsig, pi);
    dx[i] = LOAD(gamma, pi) * LOAD(rsig, pi) *
        (LOAD(dy, i) - fma(x_norm, LOAD(dgamma, pi), LOAD(dbeta, pi)) * denom);
  }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormWGrad(
    const int N,
    const int C,
    const int S,
    const Tx* x,
    const Tp* mu,
    const Tp* rsig,
    const Tx* dy,
    Tp* dgamma,
    Tp* dbeta) {
  const int outer_dim = N * S;
  __shared__ typename BlockReduce<Tp>::TempStorage dg_storage;
  __shared__ typename BlockReduce<Tp>::TempStorage db_storage;
  CUDA_2D_KERNEL_LOOP1(i, C) {
    Tp dg_val = Tp(0), db_val = Tp(0);
    CUDA_2D_KERNEL_LOOP2(j, outer_dim) {
      const int xi = kOrder == StorageOrder::NCHW ? (j / S * C + i) * S + j % S
                                                  : j * C + i;
      dg_val += LOAD(dy, xi) * (LOAD(x, xi) - LOAD(mu, i)) * LOAD(rsig, i);
      db_val += LOAD(dy, xi);
    }
    dg_val = BlockReduce<Tp>(dg_storage).Reduce(dg_val, cub::Sum());
    db_val = BlockReduce<Tp>(db_storage).Reduce(db_val, cub::Sum());
    if (threadIdx.x == 0) {
      dgamma[i] = dg_val;
      dbeta[i] = db_val;
    }
  }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _BatchNormInferenceGrad(
    const int nthreads,
    const int C,
    const int S,
    const Tp* rsig,
    const Tp* gamma,
    const Tx* dy,
    Tx* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const int pi = kOrder == StorageOrder::NCHW ? (i / S) % C : i % C;
    dx[i] = LOAD(gamma, pi) * LOAD(dy, i) * LOAD(rsig, pi);
  }
}

#undef LOAD

} // namespace

/* ------------------- Launcher Separator ------------------- */

#define DEFINE_GRAD_KERNEL_LAUNCHER(Tx, Tp)                                  \
  template <>                                                                \
  void BatchNormExpectation<Tx, Tp, CUDAContext>(                            \
      const int N,                                                           \
      const int C,                                                           \
      const int S,                                                           \
      const Tp denorm,                                                       \
      const string& data_format,                                             \
      const Tx* x,                                                           \
      Tp* ex,                                                                \
      Tp* ex2,                                                               \
      CUDAContext* ctx) {                                                    \
    if (data_format == "NCHW") {                                             \
      _BatchNormExpectation<Tx, Tp, StorageOrder::NCHW>                      \
          <<<CUDA_2D_BLOCKS(C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(      \
              N, C, S, denorm, x, ex, ex2);                                  \
    } else if (data_format == "NHWC") {                                      \
      _BatchNormExpectation<Tx, Tp, StorageOrder::NHWC>                      \
          <<<CUDA_2D_BLOCKS(C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(      \
              N, C, S, denorm, x, ex, ex2);                                  \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  void BatchNormInternalGrad<Tx, Tp, CUDAContext>(                           \
      const int N,                                                           \
      const int C,                                                           \
      const int S,                                                           \
      const string& data_format,                                             \
      const Tx* x,                                                           \
      const Tp* mu,                                                          \
      const Tp* rsig,                                                        \
      const Tp* gamma,                                                       \
      const Tx* dy,                                                          \
      Tp* dgamma,                                                            \
      Tp* dbeta,                                                             \
      CUDAContext* ctx) {                                                    \
    if (data_format == "NCHW") {                                             \
      _BatchNormInternalGrad<Tx, Tp, StorageOrder::NCHW>                     \
          <<<CUDA_2D_BLOCKS(C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(      \
              N, C, S, x, mu, rsig, gamma, dy, dgamma, dbeta);               \
    } else if (data_format == "NHWC") {                                      \
      _BatchNormInternalGrad<Tx, Tp, StorageOrder::NHWC>                     \
          <<<CUDA_2D_BLOCKS(C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(      \
              N, C, S, x, mu, rsig, gamma, dy, dgamma, dbeta);               \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  void BatchNormTrainingGrad<Tx, Tp, CUDAContext>(                           \
      const int N,                                                           \
      const int C,                                                           \
      const int S,                                                           \
      const string& data_format,                                             \
      const Tx* x,                                                           \
      const Tp* mu,                                                          \
      const Tp* rsig,                                                        \
      const Tp* gamma,                                                       \
      const Tp* dgamma,                                                      \
      const Tp* dbeta,                                                       \
      const Tx* dy,                                                          \
      Tx* dx,                                                                \
      CUDAContext* ctx) {                                                    \
    const int nthreads = N * C * S;                                          \
    if (data_format == "NCHW") {                                             \
      _BatchNormTrainingGrad<Tx, Tp, StorageOrder::NCHW>                     \
          <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              nthreads, N, C, S, x, mu, rsig, gamma, dgamma, dbeta, dy, dx); \
    } else if (data_format == "NHWC") {                                      \
      _BatchNormTrainingGrad<Tx, Tp, StorageOrder::NHWC>                     \
          <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              nthreads, N, C, S, x, mu, rsig, gamma, dgamma, dbeta, dy, dx); \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  void BatchNormBackwardTraining<Tx, Tp, CUDAContext>(                       \
      const int N,                                                           \
      const int C,                                                           \
      const int S,                                                           \
      const string& data_format,                                             \
      const Tx* x,                                                           \
      const Tp* mu,                                                          \
      const Tp* rsig,                                                        \
      const Tp* gamma,                                                       \
      const Tx* dy,                                                          \
      Tx* dx,                                                                \
      Tp* dgamma,                                                            \
      Tp* dbeta,                                                             \
      CUDAContext* ctx) {                                                    \
    const int nthreads = N * C * S;                                          \
    if (data_format == "NCHW") {                                             \
      _BatchNormInternalGrad<Tx, Tp, StorageOrder::NCHW>                     \
          <<<CUDA_2D_BLOCKS(C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(      \
              N, C, S, x, mu, rsig, gamma, dy, dgamma, dbeta);               \
      _BatchNormTrainingGrad<Tx, Tp, StorageOrder::NCHW>                     \
          <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              nthreads, N, C, S, x, mu, rsig, gamma, dgamma, dbeta, dy, dx); \
    } else if (data_format == "NHWC") {                                      \
      _BatchNormInternalGrad<Tx, Tp, StorageOrder::NHWC>                     \
          <<<CUDA_2D_BLOCKS(C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(      \
              N, C, S, x, mu, rsig, gamma, dy, dgamma, dbeta);               \
      _BatchNormTrainingGrad<Tx, Tp, StorageOrder::NHWC>                     \
          <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              nthreads, N, C, S, x, mu, rsig, gamma, dgamma, dbeta, dy, dx); \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  void BatchNormBackwardInference<Tx, Tp, CUDAContext>(                      \
      const int N,                                                           \
      const int C,                                                           \
      const int S,                                                           \
      const string& data_format,                                             \
      const Tx* x,                                                           \
      const Tp* mu,                                                          \
      const Tp* rsig,                                                        \
      const Tp* gamma,                                                       \
      const Tx* dy,                                                          \
      Tx* dx,                                                                \
      Tp* dgamma,                                                            \
      Tp* dbeta,                                                             \
      CUDAContext* ctx) {                                                    \
    const int nthreads = N * C * S;                                          \
    if (data_format == "NCHW") {                                             \
      if (dgamma != nullptr) {                                               \
        _BatchNormWGrad<Tx, Tp, StorageOrder::NCHW>                          \
            <<<CUDA_2D_BLOCKS(C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(    \
                N, C, S, x, mu, rsig, dy, dgamma, dbeta);                    \
      }                                                                      \
      _BatchNormInferenceGrad<Tx, Tp, StorageOrder::NCHW>                    \
          <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              nthreads, C, S, rsig, gamma, dy, dx);                          \
    } else if (data_format == "NHWC") {                                      \
      if (dgamma != nullptr) {                                               \
        _BatchNormWGrad<Tx, Tp, StorageOrder::NHWC>                          \
            <<<CUDA_2D_BLOCKS(C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(    \
                N, C, S, x, mu, rsig, dy, dgamma, dbeta);                    \
      }                                                                      \
      _BatchNormInferenceGrad<Tx, Tp, StorageOrder::NHWC>                    \
          <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              nthreads, C, S, rsig, gamma, dy, dx);                          \
    }                                                                        \
  }

DEFINE_GRAD_KERNEL_LAUNCHER(float, float);

#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
