
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/memory.h"
#include "dragon/utils/device/common_cub.h"
#include "dragon/utils/math_functions.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

#if __CUDA_ARCH__ >= 350
#define LOAD(x, i) __ldg(x + i)
#define LOADF(x, i) __half2float(__ldg(x + i))
#else
#define LOAD(x, i) x[i]
#define LOADF(x, i) __half2float(x[i])
#endif

namespace {

template <typename T>
__global__ void _GroupNormFusedParams(
    const int N,
    const int G,
    const int D,
    const T* mu,
    const T* rsig,
    const T* gamma,
    const T* beta,
    T* scale,
    T* bias) {
  const int outer_dim = N * G;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    const int g = i % G;
    const T mu_val = LOAD(mu, i);
    const T rsig_val = LOAD(rsig, i);
    CUDA_2D_KERNEL_LOOP2(j, D) {
      const int wi = i * D + j;
      const int gi = g * D + j;
      const T w = LOAD(gamma, gi) * rsig_val;
      scale[wi] = w;
      bias[wi] = fma(-w, mu_val, LOAD(beta, gi));
    }
  }
}

template <typename Tx, typename Tp>
__global__ void _GroupNormForwardNCHW(
    const int N,
    const int C,
    const int S,
    const Tx* x,
    const Tp* scale,
    const Tp* bias,
    Tx* y) {
  const int outer_dim = N * C;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    const Tp w = LOAD(scale, i);
    const Tp b = LOAD(bias, i);
    CUDA_2D_KERNEL_LOOP2(j, S) {
      const int xi = i * S + j;
      y[xi] = fma(LOAD(x, xi), w, b);
    }
  }
}

template <>
__global__ void _GroupNormForwardNCHW<half, float>(
    const int N,
    const int C,
    const int S,
    const half* x,
    const float* scale,
    const float* bias,
    half* y) {
  const int outer_dim = N * C;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    const float w = LOAD(scale, i);
    const float b = LOAD(bias, i);
    CUDA_2D_KERNEL_LOOP2(j, S) {
      const int xi = i * S + j;
      y[xi] = __float2half(fmaf(LOADF(x, xi), w, b));
    }
  }
}

template <typename Tx, typename Tp>
__global__ void _GroupNormForwardNHWC(
    const int N,
    const int C,
    const int S,
    const Tx* x,
    const Tp* scale,
    const Tp* bias,
    Tx* y) {
  const int outer_dim = N * S;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    const int n = i / S;
    CUDA_2D_KERNEL_LOOP2(j, C) {
      const int xi = i * C + j;
      const int wi = n * C + j;
      y[xi] = fma(LOAD(x, xi), LOAD(scale, wi), LOAD(bias, wi));
    }
  }
}

template <>
__global__ void _GroupNormForwardNHWC<half, float>(
    const int N,
    const int C,
    const int S,
    const half* x,
    const float* scale,
    const float* bias,
    half* y) {
  const int outer_dim = N * S;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    const int n = i / S;
    CUDA_2D_KERNEL_LOOP2(j, C) {
      const int xi = i * C + j;
      const int wi = n * C + j;
      y[xi] = __float2half(fmaf(LOADF(x, xi), LOAD(scale, wi), LOAD(bias, wi)));
    }
  }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _GroupNormWGrad(
    const int N,
    const int G,
    const int D,
    const int S,
    const Tx* x,
    const Tp* mu,
    const Tp* rsig,
    const Tx* dy,
    Tp* dgamma,
    Tp* dbeta) {
  const int outer_dim = G * D;
  const int inner_dim = N * S;
  __shared__ typename BlockReduce<Tp>::TempStorage dg_storage;
  __shared__ typename BlockReduce<Tp>::TempStorage db_storage;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    Tp dg_val = Tp(0), db_val = Tp(0);
    CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
      const int n = j / S;
      const int xi = kOrder == StorageOrder::NCHW
          ? (n * outer_dim + i) * S + j % S
          : j * outer_dim + i;
      const int mi = n * G + i / D;
      dg_val += LOAD(dy, xi) * (LOAD(x, xi) - LOAD(mu, mi)) * LOAD(rsig, mi);
      db_val += LOAD(dy, xi);
    }
    dg_val = BlockReduce<Tp>(dg_storage).Reduce(dg_val, cub::Sum());
    db_val = BlockReduce<Tp>(db_storage).Reduce(db_val, cub::Sum());
    if (threadIdx.x == 0) {
      dgamma[i] = dg_val;
      dbeta[i] = db_val;
    }
  }
}

template <StorageOrder kOrder>
__global__ void _GroupNormWGradHalf(
    const int N,
    const int G,
    const int D,
    const int S,
    const half* x,
    const float* mu,
    const float* rsig,
    const half* dy,
    float* dgamma,
    float* dbeta) {
  const int outer_dim = G * D;
  const int inner_dim = N * S;
  __shared__ typename BlockReduce<float>::TempStorage dg_storage;
  __shared__ typename BlockReduce<float>::TempStorage db_storage;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    float dg_val = 0.f, db_val = 0.f;
    CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
      const int n = j / S;
      const int xi = kOrder == StorageOrder::NCHW
          ? (n * outer_dim + i) * S + j % S
          : j * outer_dim + i;
      const int mi = n * G + i / D;
      dg_val += LOADF(dy, xi) * (LOADF(x, xi) - LOAD(mu, mi)) * LOAD(rsig, mi);
      db_val += LOADF(dy, xi);
    }
    dg_val = BlockReduce<float>(dg_storage).Reduce(dg_val, cub::Sum());
    db_val = BlockReduce<float>(db_storage).Reduce(db_val, cub::Sum());
    if (threadIdx.x == 0) {
      dgamma[i] = dg_val;
      dbeta[i] = db_val;
    }
  }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _GroupNormInternalGrad(
    const int N,
    const int G,
    const int D,
    const int S,
    const Tx* x,
    const Tp* gamma,
    const Tx* dy,
    Tp* ds,
    Tp* db) {
  const int outer_dim = N * G;
  const int inner_dim = D * S;
  __shared__ typename BlockReduce<Tp>::TempStorage ds_storage;
  __shared__ typename BlockReduce<Tp>::TempStorage db_storage;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    Tp ds_val = Tp(0), db_val = Tp(0);
    CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
      const int gi = i % G * D + j / S;
      const int xi = kOrder == StorageOrder::NCHW
          ? i * inner_dim + j
          : (i / G * S + j % S) * G * D + gi;
      ds_val += LOAD(gamma, gi) * LOAD(dy, xi) * LOAD(x, xi);
      db_val += LOAD(gamma, gi) * LOAD(dy, xi);
    }
    ds_val = BlockReduce<Tp>(ds_storage).Reduce(ds_val, cub::Sum());
    db_val = BlockReduce<Tp>(db_storage).Reduce(db_val, cub::Sum());
    if (threadIdx.x == 0) {
      ds[i] = ds_val;
      db[i] = db_val;
    }
  }
}

template <StorageOrder kOrder>
__global__ void _GroupNormInternalGradHalf(
    const int N,
    const int G,
    const int D,
    const int S,
    const half* x,
    const float* gamma,
    const half* dy,
    float* ds,
    float* db) {
  const int outer_dim = N * G;
  const int inner_dim = D * S;
  __shared__ typename BlockReduce<float>::TempStorage ds_storage;
  __shared__ typename BlockReduce<float>::TempStorage db_storage;
  CUDA_2D_KERNEL_LOOP1(i, outer_dim) {
    float ds_val = 0.f, db_val = 0.f;
    CUDA_2D_KERNEL_LOOP2(j, inner_dim) {
      const int gi = i % G * D + j / S;
      const int xi = kOrder == StorageOrder::NCHW
          ? i * inner_dim + j
          : (i / G * S + j % S) * G * D + gi;
      ds_val += LOAD(gamma, gi) * LOADF(dy, xi) * LOADF(x, xi);
      db_val += LOAD(gamma, gi) * LOADF(dy, xi);
    }
    ds_val = BlockReduce<float>(ds_storage).Reduce(ds_val, cub::Sum());
    db_val = BlockReduce<float>(db_storage).Reduce(db_val, cub::Sum());
    if (threadIdx.x == 0) {
      ds[i] = ds_val;
      db[i] = db_val;
    }
  }
}

template <typename Tx, typename Tp, StorageOrder kOrder>
__global__ void _GroupNormGrad(
    const int nthreads,
    const int G,
    const int D,
    const int S,
    const Tx* x,
    const Tp* mu,
    const Tp* rsig,
    const Tp* gamma,
    const Tp* ds,
    const Tp* db,
    const Tx* dy,
    Tx* dx) {
  const int C = G * D;
  const Tp denom = Tp(1) / Tp(D * S);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const int mi = kOrder == StorageOrder::NCHW ? i / (D * S)
                                                : i / (C * S) * G + (i / D % G);
    const int gi = kOrder == StorageOrder::NCHW ? (i / S) % C : i % C;
    const Tp u = fma(LOAD(db, mi), LOAD(mu, mi), -LOAD(ds, mi)) *
        (LOAD(x, i) - LOAD(mu, mi)) * utils::math::Cube(LOAD(rsig, mi));
    const Tp v = LOAD(db, mi) * LOAD(rsig, mi);
    dx[i] = LOAD(gamma, gi) * LOAD(dy, i) * LOAD(rsig, mi) + (u - v) * denom;
  }
}

template <StorageOrder kOrder>
__global__ void _GroupNormGradHalf(
    const int nthreads,
    const int G,
    const int D,
    const int S,
    const half* x,
    const float* mu,
    const float* rsig,
    const float* gamma,
    const float* ds,
    const float* db,
    const half* dy,
    half* dx) {
  const int C = G * D;
  const float denom = 1.f / float(D * S);
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const int mi = kOrder == StorageOrder::NCHW ? i / (D * S)
                                                : i / (C * S) * G + (i / D % G);
    const int gi = kOrder == StorageOrder::NCHW ? (i / S) % C : i % C;
    const float u = fmaf(LOAD(db, mi), LOAD(mu, mi), -LOAD(ds, mi)) *
        (LOADF(x, i) - LOAD(mu, mi)) * utils::math::Cube(LOAD(rsig, mi));
    const float v = LOAD(db, mi) * LOAD(rsig, mi);
    dx[i] = __float2half(
        LOAD(gamma, gi) * LOADF(dy, i) * LOAD(rsig, mi) + (u - v) * denom);
  }
}

#undef LOAD
#undef LOADF

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void GroupNormForward<float16, float, CUDAContext>(
    const int N,
    const int G,
    const int D,
    const int S,
    const string& data_format,
    const float16* x,
    const float* mu,
    const float* rsig,
    const float* gamma,
    const float* beta,
    float* scale,
    float* bias,
    float16* y,
    CUDAContext* ctx) {
  const int C = G * D;
  _GroupNormFusedParams<float>
      <<<CUDA_2D_BLOCKS(N * G), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
          N, G, D, mu, rsig, gamma, beta, scale, bias);
  if (data_format == "NCHW") {
    _GroupNormForwardNCHW<half, float>
        <<<CUDA_2D_BLOCKS(N * C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
            N,
            C,
            S,
            reinterpret_cast<const half*>(x),
            scale,
            bias,
            reinterpret_cast<half*>(y));
  } else if (data_format == "NHWC") {
    _GroupNormForwardNHWC<half, float>
        <<<CUDA_2D_BLOCKS(N * C), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
            N,
            C,
            S,
            reinterpret_cast<const half*>(x),
            scale,
            bias,
            reinterpret_cast<half*>(y));
  }
}

template <>
void GroupNormBackward<float16, float, CUDAContext>(
    const int N,
    const int G,
    const int D,
    const int S,
    const string& data_format,
    const float16* x,
    const float* mu,
    const float* rsig,
    const float* gamma,
    const float16* dy,
    float* ds,
    float* db,
    float16* dx,
    float* dgamma,
    float* dbeta,
    CUDAContext* ctx) {
  auto nthreads = N * G * D * S;
  if (data_format == "NCHW") {
    _GroupNormWGradHalf<StorageOrder::NCHW>
        <<<CUDA_2D_BLOCKS(G * D), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
            N,
            G,
            D,
            S,
            reinterpret_cast<const half*>(x),
            mu,
            rsig,
            reinterpret_cast<const half*>(dy),
            dgamma,
            dbeta);
    _GroupNormInternalGradHalf<StorageOrder::NCHW>
        <<<CUDA_2D_BLOCKS(N * G), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
            N,
            G,
            D,
            S,
            reinterpret_cast<const half*>(x),
            gamma,
            reinterpret_cast<const half*>(dy),
            ds,
            db);
    _GroupNormGradHalf<StorageOrder::NCHW>
        <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
            nthreads,
            G,
            D,
            S,
            reinterpret_cast<const half*>(x),
            mu,
            rsig,
            gamma,
            ds,
            db,
            reinterpret_cast<const half*>(dy),
            reinterpret_cast<half*>(dx));
  } else if (data_format == "NHWC") {
    _GroupNormWGradHalf<StorageOrder::NHWC>
        <<<CUDA_2D_BLOCKS(G * D), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
            N,
            G,
            D,
            S,
            reinterpret_cast<const half*>(x),
            mu,
            rsig,
            reinterpret_cast<const half*>(dy),
            dgamma,
            dbeta);
    _GroupNormInternalGradHalf<StorageOrder::NHWC>
        <<<CUDA_2D_BLOCKS(N * G), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
            N,
            G,
            D,
            S,
            reinterpret_cast<const half*>(x),
            gamma,
            reinterpret_cast<const half*>(dy),
            ds,
            db);
    _GroupNormGradHalf<StorageOrder::NHWC>
        <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
            nthreads,
            G,
            D,
            S,
            reinterpret_cast<const half*>(x),
            mu,
            rsig,
            gamma,
            ds,
            db,
            reinterpret_cast<const half*>(dy),
            reinterpret_cast<half*>(dx));
  }
} // GroupNormBackward

#define DEFINE_KERNEL_LAUNCHER(Tx, Tp)                                     \
  template <>                                                              \
  void GroupNormForward<Tx, Tp, CUDAContext>(                              \
      const int N,                                                         \
      const int G,                                                         \
      const int D,                                                         \
      const int S,                                                         \
      const string& data_format,                                           \
      const Tx* x,                                                         \
      const Tp* mu,                                                        \
      const Tp* rsig,                                                      \
      const Tp* gamma,                                                     \
      const Tp* beta,                                                      \
      Tp* scale,                                                           \
      Tp* bias,                                                            \
      Tx* y,                                                               \
      CUDAContext* ctx) {                                                  \
    const int C = G * D;                                                   \
    _GroupNormFusedParams<Tp>                                              \
        <<<CUDA_2D_BLOCKS(N* G), CUDA_THREADS, 0, ctx->cuda_stream()>>>(   \
            N, G, D, mu, rsig, gamma, beta, scale, bias);                  \
    if (data_format == "NCHW") {                                           \
      _GroupNormForwardNCHW<Tx, Tp>                                        \
          <<<CUDA_2D_BLOCKS(N* C), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
              N, C, S, x, scale, bias, y);                                 \
    } else if (data_format == "NHWC") {                                    \
      _GroupNormForwardNHWC<Tx, Tp>                                        \
          <<<CUDA_2D_BLOCKS(N* C), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
              N, C, S, x, scale, bias, y);                                 \
    }                                                                      \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(Tx, Tp)                                 \
  template <>                                                               \
  void GroupNormBackward<Tx, Tp, CUDAContext>(                              \
      const int N,                                                          \
      const int G,                                                          \
      const int D,                                                          \
      const int S,                                                          \
      const string& data_format,                                            \
      const Tx* x,                                                          \
      const Tp* mu,                                                         \
      const Tp* rsig,                                                       \
      const Tp* gamma,                                                      \
      const Tx* dy,                                                         \
      Tp* ds,                                                               \
      Tp* db,                                                               \
      Tx* dx,                                                               \
      Tp* dgamma,                                                           \
      Tp* dbeta,                                                            \
      CUDAContext* ctx) {                                                   \
    auto nthreads = N * G * D * S;                                          \
    if (data_format == "NCHW") {                                            \
      _GroupNormWGrad<Tx, Tp, StorageOrder::NCHW>                           \
          <<<CUDA_2D_BLOCKS(G* D), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              N, G, D, S, x, mu, rsig, dy, dgamma, dbeta);                  \
      _GroupNormInternalGrad<Tx, Tp, StorageOrder::NCHW>                    \
          <<<CUDA_2D_BLOCKS(N* G), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              N, G, D, S, x, gamma, dy, ds, db);                            \
      _GroupNormGrad<Tx, Tp, StorageOrder::NCHW>                            \
          <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
              nthreads, G, D, S, x, mu, rsig, gamma, ds, db, dy, dx);       \
    } else if (data_format == "NHWC") {                                     \
      _GroupNormWGrad<Tx, Tp, StorageOrder::NHWC>                           \
          <<<CUDA_2D_BLOCKS(G* D), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              N, G, D, S, x, mu, rsig, dy, dgamma, dbeta);                  \
      _GroupNormInternalGrad<Tx, Tp, StorageOrder::NHWC>                    \
          <<<CUDA_2D_BLOCKS(N* G), CUDA_THREADS, 0, ctx->cuda_stream()>>>(  \
              N, G, D, S, x, gamma, dy, ds, db);                            \
      _GroupNormGrad<Tx, Tp, StorageOrder::NHWC>                            \
          <<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
              nthreads, G, D, S, x, mu, rsig, gamma, ds, db, dy, dx);       \
    }                                                                       \
  }

DEFINE_KERNEL_LAUNCHER(float, float);
DEFINE_GRAD_KERNEL_LAUNCHER(float, float);

#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
