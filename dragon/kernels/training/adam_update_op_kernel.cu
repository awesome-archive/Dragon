
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T>
__global__ void _AdamUpdate(
    const int nthreads,
    const T lr,
    const T beta1,
    const T beta2,
    const T eps,
    T* g,
    T* m,
    T* v) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    T gi = g[i];
    T mi = m[i] = m[i] * beta1 + gi * (1 - beta1);
    T vi = v[i] = v[i] * beta2 + gi * gi * (1 - beta2);
    g[i] = lr * mi / (sqrt(vi) + eps);
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void AdamUpdate<float, CUDAContext>(
    const int count,
    const float lr,
    const float beta1,
    const float beta2,
    const float eps,
    float* g,
    float* m,
    float* v,
    CUDAContext* ctx) {
  _AdamUpdate<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      count, lr, beta1, beta2, eps, g, m, v);
}

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
