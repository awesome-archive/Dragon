
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernels {

namespace {

template <typename T>
__global__ void _AdamUpdate(
    const int N,
    const T lr,
    const T beta1,
    const T beta2,
    const T eps,
    T* g,
    T* m,
    T* v) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    T gi = g[i];
    T mi = m[i] = m[i] * beta1 + gi * (1 - beta1);
    T vi = v[i] = v[i] * beta2 + gi * gi * (1 - beta2);
    g[i] = lr * mi / (sqrt(vi) + eps);
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void AdamUpdate<float, CUDAContext>(
    const int N,
    const float lr,
    const float beta1,
    const float beta2,
    const float eps,
    float* g,
    float* m,
    float* v,
    CUDAContext* ctx) {
  _AdamUpdate<<<CUDA_BLOCKS(N), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      N, lr, beta1, beta2, eps, g, m, v);
}

} // namespace kernels

} // namespace dragon

#endif // USE_CUDA
