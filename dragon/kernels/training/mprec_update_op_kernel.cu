
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

__global__ void _MixedPrecL2DecayHalf(
    const int nthreads,
    const float alpha,
    const half* w,
    float* dx) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    dx[i] += __half2float(w[i]) * alpha;
#endif
  }
}

__global__ void
_MixedPrecUpdateHalf(const int nthreads, const float* updates, half* w) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
#if __CUDA_ARCH__ >= 530
    w[i] = __float2half(__half2float(w[i]) - updates[i]);
#endif
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void MixedPrecL2Decay<float16, CUDAContext>(
    const int count,
    const float alpha,
    const float16* w,
    float* dx,
    CUDAContext* ctx) {
  _MixedPrecL2DecayHalf<<<
      CUDA_BLOCKS(count),
      CUDA_THREADS,
      0,
      ctx->cuda_stream()>>>(count, alpha, reinterpret_cast<const half*>(w), dx);
}

template <>
void MixedPrecUpdate<float16, CUDAContext>(
    const int count,
    const float* updates,
    float16* w,
    CUDAContext* ctx) {
  _MixedPrecUpdateHalf<<<
      CUDA_BLOCKS(count),
      CUDA_THREADS,
      0,
      ctx->cuda_stream()>>>(count, updates, reinterpret_cast<half*>(w));
}

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
