
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T>
__global__ void
_NesterovUpdate(const int nthreads, const T lr, const T momentum, T* g, T* m) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    T mi = m[i];
    T mi_new = m[i] = momentum * mi + lr * g[i];
    g[i] = (1 + momentum) * mi_new - momentum * mi;
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void NesterovUpdate<float, CUDAContext>(
    const int count,
    const float lr,
    const float momentum,
    float* g,
    float* m,
    CUDAContext* ctx) {
  _NesterovUpdate<<<CUDA_BLOCKS(count), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      count, lr, momentum, g, m);
}

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
