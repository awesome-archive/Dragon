
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include "dragon/core/context_cuda.h"
#include "dragon/utils/op_kernels.h"

namespace dragon {

namespace kernel {

namespace {

template <typename T>
__global__ void _RoiPool(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const float spatial_scale,
    const T* x,
    const float* rois,
    int* mask,
    T* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int ow = yi % out_w;
    const int oh = (yi / out_w) % out_h;
    const int c = (yi / out_w / out_h) % C;
    const int n = yi / out_w / out_h / C;

    const float* roi = rois + n * 5;
    const int batch_ind = roi[0];

    if (batch_ind < 0) {
      y[yi] = T(0);
      mask[yi] = -1;
      continue;
    }

    const int roi_start_w = round(roi[1] * spatial_scale);
    const int roi_start_h = round(roi[2] * spatial_scale);
    const int roi_end_w = round(roi[3] * spatial_scale);
    const int roi_end_h = round(roi[4] * spatial_scale);

    const int roi_w = max(roi_end_w - roi_start_w + 1, 1);
    const int roi_h = max(roi_end_h - roi_start_h + 1, 1);
    const float bin_h = (float)roi_h / (float)out_h;
    const float bin_w = (float)roi_w / (float)out_w;

    int hstart = floor(bin_h * oh);
    int wstart = floor(bin_w * ow);
    int hend = ceil(bin_h * (oh + 1));
    int wend = ceil(bin_w * (ow + 1));

    hstart = min(max(hstart + roi_start_h, 0), H);
    hend = min(max(hend + roi_start_h, 0), H);
    wstart = min(max(wstart + roi_start_w, 0), W);
    wend = min(max(wend + roi_start_w, 0), W);
    const bool empty = (hend <= hstart) || (wend <= wstart);

    int max_idx = empty ? -1 : 0;
    const T* offset_x = x + (batch_ind * C + c) * H * W;
    T val = empty ? T(0) : offset_x[0];

    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        const int xi = h * W + w;
#if __CUDA_ARCH__ >= 350
        if (__ldg(offset_x + xi) > val) {
          val = __ldg(offset_x + xi);
          max_idx = xi;
        }
#else
        if (offset_x[xi] > val) {
          val = offset_x[xi];
          max_idx = xi;
        }
#endif
      }
    }

    y[yi] = val;
    mask[yi] = max_idx;
  }
}

template <>
__global__ void _RoiPool<half>(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const float spatial_scale,
    const half* x,
    const float* rois,
    int* mask,
    half* y) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int ow = yi % out_w;
    const int oh = (yi / out_w) % out_h;
    const int c = (yi / out_w / out_h) % C;
    const int n = yi / out_w / out_h / C;

    const float* roi = rois + n * 5;
    const int batch_ind = roi[0];

    if (batch_ind < 0) {
      y[yi] = __float2half(0.f);
      mask[yi] = -1;
      continue;
    }

    const int roi_start_w = round(roi[1] * spatial_scale);
    const int roi_start_h = round(roi[2] * spatial_scale);
    const int roi_end_w = round(roi[3] * spatial_scale);
    const int roi_end_h = round(roi[4] * spatial_scale);

    const int roi_w = max(roi_end_w - roi_start_w + 1, 1);
    const int roi_h = max(roi_end_h - roi_start_h + 1, 1);
    const float bin_h = (float)roi_h / (float)out_h;
    const float bin_w = (float)roi_w / (float)out_w;

    int hstart = floor(bin_h * oh);
    int wstart = floor(bin_w * ow);
    int hend = ceil(bin_h * (oh + 1));
    int wend = ceil(bin_w * (ow + 1));

    hstart = min(max(hstart + roi_start_h, 0), H);
    hend = min(max(hend + roi_start_h, 0), H);
    wstart = min(max(wstart + roi_start_w, 0), W);
    wend = min(max(wend + roi_start_w, 0), W);
    const bool empty = (hend <= hstart) || (wend <= wstart);

    int max_idx = empty ? -1 : 0;
    const half* offset_x = x + ((batch_ind * C + c) * H * W);
#if __CUDA_ARCH__ >= 530
    half val = empty ? __float2half(0.f) : __ldg(offset_x);
#else
    float val = empty ? 0.f : __half2float(*offset_x);
#endif

    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        const int xi = h * W + w;
#if __CUDA_ARCH__ >= 530
        if (__hgt(__ldg(offset_x + xi), val)) {
          val = __ldg(offset_x + xi);
          max_idx = xi;
        }
#elif __CUDA_ARCH__ >= 350
        if (__half2float(__ldg(offset_x + xi)) > val) {
          val = __half2float(__ldg(offset_x + xi));
          max_idx = xi;
        }
#else
        if (__half2float(offset_x[xi]) > val) {
          val = __half2float(offset_x[xi]);
          max_idx = xi;
        }
#endif
      }
    }

#if __CUDA_ARCH__ >= 530
    y[yi] = val;
#else
    y[yi] = __float2half(val);
#endif
    mask[yi] = max_idx;
  }
}

template <typename T>
__global__ void _RoiPoolGrad(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const float spatial_scale,
    const T* dy,
    const float* rois,
    const int* mask,
    float* dx) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int c = (yi / out_w / out_h) % C;
    const int n = yi / out_w / out_h / C;

    const float* roi = rois + n * 5;
    const int batch_ind = roi[0];
    if (batch_ind < 0) continue;

    float* offset_dx = dx + (batch_ind * C + c) * H * W;
#if __CUDA_ARCH__ >= 350
    if (__ldg(mask + yi) != -1) {
      atomicAdd(offset_dx + __ldg(mask + yi), (float)dy[yi]);
    }
#else
    if (mask[yi] != -1) {
      atomicAdd(offset_dx + mask[yi], (float)dy[yi]);
    }
#endif
  }
}

template <>
__global__ void _RoiPoolGrad<half>(
    const int nthreads,
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const float spatial_scale,
    const half* dy,
    const float* rois,
    const int* mask,
    float* dx) {
  CUDA_1D_KERNEL_LOOP(yi, nthreads) {
    const int c = (yi / out_w / out_h) % C;
    const int n = yi / out_w / out_h / C;
    const float* roi = rois + n * 5;

    const int batch_ind = roi[0];
    if (batch_ind < 0) continue;

    float* offset_dx = dx + (batch_ind * C + c) * H * W;
#if __CUDA_ARCH__ >= 350
    if (__ldg(mask + yi) != -1) {
      atomicAdd(offset_dx + __ldg(mask + yi), __half2float(dy[yi]));
    }
#else
    if (mask[yi] != -1) {
      atomicAdd(offset_dx + mask[yi], __half2float(dy[yi]));
    }
#endif
  }
}

} // namespace

/* ------------------- Launcher Separator ------------------- */

template <>
void RoiPool<float16, CUDAContext>(
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const int num_rois,
    const float spatial_scale,
    const float16* x,
    const float* rois,
    int* mask,
    float16* y,
    CUDAContext* ctx) {
  auto nthreads = num_rois * C * out_h * out_w;
  _RoiPool<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      nthreads,
      C,
      H,
      W,
      out_h,
      out_w,
      spatial_scale,
      reinterpret_cast<const half*>(x),
      rois,
      mask,
      reinterpret_cast<half*>(y));
}

template <>
void RoiPoolGrad<float16, CUDAContext>(
    const int C,
    const int H,
    const int W,
    const int out_h,
    const int out_w,
    const int num_rois,
    const float spatial_scale,
    const float16* dy,
    const float* rois,
    const int* mask,
    float* dx,
    CUDAContext* ctx) {
  auto nthreads = num_rois * C * out_h * out_w;
  _RoiPoolGrad<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>(
      nthreads,
      C,
      H,
      W,
      out_h,
      out_w,
      spatial_scale,
      reinterpret_cast<const half*>(dy),
      rois,
      mask,
      dx);
} // RoiPoolGrad

#define DEFINE_KERNEL_LAUNCHER(T)                                             \
  template <>                                                                 \
  void RoiPool<T, CUDAContext>(                                               \
      const int C,                                                            \
      const int H,                                                            \
      const int W,                                                            \
      const int out_h,                                                        \
      const int out_w,                                                        \
      const int num_rois,                                                     \
      const float spatial_scale,                                              \
      const T* x,                                                             \
      const float* rois,                                                      \
      int* mask,                                                              \
      T* y,                                                                   \
      CUDAContext* ctx) {                                                     \
    auto nthreads = num_rois * C * out_h * out_w;                             \
    _RoiPool<<<CUDA_BLOCKS(nthreads), CUDA_THREADS, 0, ctx->cuda_stream()>>>( \
        nthreads, C, H, W, out_h, out_w, spatial_scale, x, rois, mask, y);    \
  }

#define DEFINE_GRAD_KERNEL_LAUNCHER(T)                                       \
  template <>                                                                \
  void RoiPoolGrad<T, CUDAContext>(                                          \
      const int C,                                                           \
      const int H,                                                           \
      const int W,                                                           \
      const int out_h,                                                       \
      const int out_w,                                                       \
      const int num_rois,                                                    \
      const float spatial_scale,                                             \
      const T* dy,                                                           \
      const float* rois,                                                     \
      const int* mask,                                                       \
      float* dx,                                                             \
      CUDAContext* ctx) {                                                    \
    auto nthreads = num_rois * C * out_h * out_w;                            \
    _RoiPoolGrad<<<                                                          \
        CUDA_BLOCKS(nthreads),                                               \
        CUDA_THREADS,                                                        \
        0,                                                                   \
        ctx->cuda_stream()>>>(                                               \
        nthreads, C, H, W, out_h, out_w, spatial_scale, dy, rois, mask, dx); \
  }

DEFINE_KERNEL_LAUNCHER(float);
DEFINE_KERNEL_LAUNCHER(double);

DEFINE_GRAD_KERNEL_LAUNCHER(float);
DEFINE_GRAD_KERNEL_LAUNCHER(double);

#undef DEFINE_KERNEL_LAUNCHER
#undef DEFINE_GRAD_KERNEL_LAUNCHER

} // namespace kernel

} // namespace dragon

#endif // USE_CUDA
